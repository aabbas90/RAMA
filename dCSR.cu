#include "hip/hip_runtime.h"
#include "dCSR.h"
#include <thrust/transform.h>
#include <thrust/tuple.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <ECLgraph.h>
#include "time_measure_util.h"
#include "utils.h"
#include <Multiply.h>
#include <CSR.h>

void dCSR::print() const
{
    assert(rows() == row_offsets.size()-1);
    assert(col_ids.size() == data.size());
    std::cout << "dimension = " << rows() << "," << cols() << "\n";
    for(size_t i=0; i<rows(); ++i)
        for(size_t l=row_offsets[i]; l<row_offsets[i+1]; ++l)
            std::cout << i << ", " << col_ids[l] << ", " << data[l] << "\n"; 
}

void dCSR::compare(const dCSR& mat) const
{
    float tol = 1.0;
    assert(rows() == mat.rows());
    assert(cols() == mat.cols());
    assert(nnz() == mat.nnz());
    thrust::host_vector<int> row_offsets_h = row_offsets;
    thrust::host_vector<int> mat_row_offsets_h = mat.row_offsets;
    thrust::host_vector<int> col_ids_h = col_ids;
    thrust::host_vector<int> mat_col_ids_h = mat.col_ids;
    thrust::host_vector<int> data_h = data;
    thrust::host_vector<int> mat_data_h = mat.data;
    for(size_t i=0; i<rows(); ++i)
    {
        assert(row_offsets_h[i] == mat_row_offsets_h[i]);
        assert(row_offsets_h[i + 1] == mat_row_offsets_h[i + 1]);

        for(size_t l=row_offsets_h[i]; l<row_offsets_h[i+1]; ++l)
        {
            assert(col_ids_h[l] == mat_col_ids_h[l]);
            
            assert(std::abs(data_h[l] / mat_data_h[l] - 1) > 0.01);
            // if(std::abs(data_h[l] - mat_data_h[l]) > tol)
            // {
            //     std::cout<<"data_h[l]: "<<data_h[l]<<", mat_data_h[l]"<<mat_data_h[l]<<std::endl;
            //     assert(false);
            // }
        }
    }
}

dCSR dCSR::transpose(hipsparseHandle_t handle) const
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
    dCSR t;
    t.cols_ = rows();
    t.rows_ = cols();

    t.row_offsets = thrust::device_vector<int>(cols()+1);
    t.col_ids = thrust::device_vector<int>(nnz());
    t.data = thrust::device_vector<float>(nnz());

    // make buffer
    void* dbuffer = NULL;
    size_t bufferSize = 0;
    checkCuSparseError(hipsparseCsr2cscEx2_bufferSize(handle, rows(), cols(), nnz(), 
			thrust::raw_pointer_cast(data.data()), thrust::raw_pointer_cast(row_offsets.data()), thrust::raw_pointer_cast(col_ids.data()),
			thrust::raw_pointer_cast(t.data.data()), thrust::raw_pointer_cast(t.row_offsets.data()), thrust::raw_pointer_cast(t.col_ids.data()), 
            HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &bufferSize), "transpose buffer failed");
    
    checkCudaError(hipMalloc((void**) &dbuffer, bufferSize), "transpose buffer allocation failed");

    checkCuSparseError(hipsparseCsr2cscEx2(handle, rows(), cols(), nnz(), 
			thrust::raw_pointer_cast(data.data()), thrust::raw_pointer_cast(row_offsets.data()), thrust::raw_pointer_cast(col_ids.data()),
			thrust::raw_pointer_cast(t.data.data()), thrust::raw_pointer_cast(t.row_offsets.data()), thrust::raw_pointer_cast(t.col_ids.data()), 
            HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, dbuffer),
            "transpose failed");

    hipFree(dbuffer);
    return t;
}

template <typename T>
struct non_zero_indicator_func
{
    const T _tol;
    non_zero_indicator_func(T tol): _tol(tol) {} 

    __host__ __device__
        bool operator()(const thrust::tuple<int,int,float> t)
        {
            if(fabs(thrust::get<2>(t)) >= _tol)
                return false;
            else
                return true;
        }
};

void dCSR::compress(hipsparseHandle_t handle, const float tol)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
    thrust::device_vector<int> _row_ids = row_ids(handle);
    
    auto first = thrust::make_zip_iterator(thrust::make_tuple(col_ids.begin(), _row_ids.begin(), data.begin()));
    auto last = thrust::make_zip_iterator(thrust::make_tuple(col_ids.end(), _row_ids.end(), data.end()));

    auto new_last = thrust::remove_if(first, last, non_zero_indicator_func<float>(tol));

    const size_t nr_non_zeros = std::distance(first, new_last);
    col_ids.resize(nr_non_zeros);
    _row_ids.resize(nr_non_zeros);
    data.resize(nr_non_zeros);

    // remove_if is stable so sorting should not be required.
    // coo_sorting(handle, col_ids, _row_ids, data);

    // // now row indices are non-decreasing
    // assert(thrust::is_sorted(_row_ids.begin(), _row_ids.end()));

    cols_ = *thrust::max_element(col_ids.begin(), col_ids.end()) + 1;
    rows_ = _row_ids.back() + 1;

    row_offsets = thrust::device_vector<int>(rows_ + 1);
    hipsparseXcoo2csr(handle, thrust::raw_pointer_cast(_row_ids.data()), nnz(), rows(), thrust::raw_pointer_cast(row_offsets.data()), HIPSPARSE_INDEX_BASE_ZERO);
}

template <typename T>
struct keep_geq
{
    const T _thresh;
    keep_geq(T thresh): _thresh(thresh) {} 
   __host__ __device__ float operator()(const T &x) const
   {
     return x >= _thresh ? x : 0;
   }
};

template <typename T>
struct is_positive
{
    __host__ __device__ bool operator()(const T &x)
    {
        return x > 0;
    }
};

dCSR dCSR::keep_top_k_positive_values(hipsparseHandle_t handle, const int top_k)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
    // Create copy of self:
    dCSR p;
    p.rows_ = rows();
    p.cols_ = cols();
    p.row_offsets = row_offsets;
    p.col_ids = col_ids;
    p.data = data;

    // Set all negatives values to zero.
    thrust::transform(p.data.begin(), p.data.end(), p.data.begin(), keep_geq<float>(0.0f));
    int num_positive = thrust::count_if(thrust::device, p.data.begin(), p.data.end(), is_positive<float>());

    if (top_k < num_positive)
    {
        thrust::device_vector<float> temp = p.data;
        thrust::sort(temp.begin(), temp.end(), thrust::greater<float>()); // Ideal would be https://github.com/NVIDIA/thrust/issues/75

        float min_value_to_keep = temp[top_k];
        thrust::transform(p.data.begin(), p.data.end(), p.data.begin(), keep_geq<float>(min_value_to_keep));
    }

    p.compress(handle);

    return p;
}

dCSR multiply_slow(hipsparseHandle_t handle, dCSR& A, dCSR& B)
{
    float alpha = 1.0;
    MEASURE_FUNCTION_EXECUTION_TIME
    assert(A.cols() == B.rows());
    dCSR C;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipsparseMatDescr_t desc;
    hipsparseCreateMatDescr(&desc);
    hipsparseSetMatType(desc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(desc, HIPSPARSE_INDEX_BASE_ZERO);

    csrgemm2Info_t info = NULL;
    hipsparseCreateCsrgemm2Info(&info);

    size_t buffer_size;
    hipsparseScsrgemm2_bufferSizeExt(handle, A.rows(), B.cols(), A.cols(), 
                                &alpha,
                                desc, A.nnz(), 
                                thrust::raw_pointer_cast(A.row_offsets.data()), 
                                thrust::raw_pointer_cast(A.col_ids.data()),
                                desc, B.nnz(),
                                thrust::raw_pointer_cast(B.row_offsets.data()), 
                                thrust::raw_pointer_cast(B.col_ids.data()),
                                NULL,
                                desc, B.nnz(), 
                                thrust::raw_pointer_cast(B.row_offsets.data()), 
                                thrust::raw_pointer_cast(B.col_ids.data()),
                                info, &buffer_size);
    void* buffer = NULL;
    hipMalloc(&buffer, buffer_size);

    // Allocate memory for C
    C.rows_ = A.rows();
    C.cols_ = B.cols();
    C.row_offsets = thrust::device_vector<int>(A.rows()+1);
    int nnzC;
    int *nnzTotalDevHostPtr = &nnzC;
    hipsparseXcsrgemm2Nnz(handle, A.rows(), B.cols(), A.cols(),
                        desc, A.nnz(),
                        thrust::raw_pointer_cast(A.row_offsets.data()), 
                        thrust::raw_pointer_cast(A.col_ids.data()),
                        desc, B.nnz(), 
                        thrust::raw_pointer_cast(B.row_offsets.data()), 
                        thrust::raw_pointer_cast(B.col_ids.data()),
                        desc, B.nnz(), 
                        thrust::raw_pointer_cast(B.row_offsets.data()), 
                        thrust::raw_pointer_cast(B.col_ids.data()),
                        desc, 
                        thrust::raw_pointer_cast(C.row_offsets.data()), 
                        nnzTotalDevHostPtr,
                        info, buffer);

    C.col_ids = thrust::device_vector<int>(nnzC);
    C.data = thrust::device_vector<float>(nnzC);

    hipsparseScsrgemm2(handle, A.rows(), B.cols(), A.cols(), &alpha,
                            desc, A.nnz(), 
                            thrust::raw_pointer_cast(A.data.data()), 
                            thrust::raw_pointer_cast(A.row_offsets.data()), 
                            thrust::raw_pointer_cast(A.col_ids.data()),
                            desc, B.nnz(), 
                            thrust::raw_pointer_cast(B.data.data()), 
                            thrust::raw_pointer_cast(B.row_offsets.data()), 
                            thrust::raw_pointer_cast(B.col_ids.data()),
                            NULL,
                            desc, B.nnz(), 
                            thrust::raw_pointer_cast(B.data.data()), 
                            thrust::raw_pointer_cast(B.row_offsets.data()), 
                            thrust::raw_pointer_cast(B.col_ids.data()),
                            desc, 
                            thrust::raw_pointer_cast(C.data.data()), 
                            thrust::raw_pointer_cast(C.row_offsets.data()), 
                            thrust::raw_pointer_cast(C.col_ids.data()),
                            info, buffer);

    hipsparseDestroyCsrgemm2Info(info);
    hipsparseDestroyMatDescr(desc);
    hipFree(buffer);

    return C;
}

dCSR multiply(hipsparseHandle_t handle, dCSR& A, dCSR& B)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
    assert(A.cols() == B.rows());
    float duration;
    dCSR C;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // CUSPARSE API 
    hipsparseSpMatDescr_t matA, matB, matC;
    float alpha = 1.0f;
    float beta = 0.0f;
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType computeType = HIP_R_32F;
    void* dBuffer1 = NULL, *dBuffer2 = NULL;
    size_t bufferSize1 = 0, bufferSize2 = 0;

    int* rp = thrust::raw_pointer_cast(A.row_offsets.data());

    checkCuSparseError(hipsparseCreateCsr(&matA, A.rows(), A.cols(), A.nnz(),
                                      thrust::raw_pointer_cast(A.row_offsets.data()), 
                                      thrust::raw_pointer_cast(A.col_ids.data()), 
                                      thrust::raw_pointer_cast(A.data.data()),
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F), "Matrix descriptor init failed");

    checkCuSparseError(hipsparseCreateCsr(&matB, B.rows(), B.cols(), B.nnz(),
                                      thrust::raw_pointer_cast(B.row_offsets.data()), 
                                      thrust::raw_pointer_cast(B.col_ids.data()), 
                                      thrust::raw_pointer_cast(B.data.data()),
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F), "Matrix descriptor init failed");

    checkCuSparseError(hipsparseCreateCsr(&matC, A.rows(), B.cols(), 0,
                                      NULL, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F), "Matrix descriptor init failed");

    // SpGEMM Computation
    // ############################
    hipEventRecord(start);
    // ############################
    
    hipsparseSpGEMMDescr_t spgemmDesc;
    checkCuSparseError(hipsparseSpGEMM_createDescr(&spgemmDesc), "sparse MM desc. failed");

    // ask bufferSize1 bytes for external memory
    checkCuSparseError(hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, NULL), "spGEMM work estimation 1 failed");

    checkCudaError(hipMalloc((void**) &dBuffer1, bufferSize1), "buffer 1 allocation failed");

    // inspect the matrices A and B to understand the memory requirement for the next step
    checkCuSparseError(hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, dBuffer1), "spGEMM work estimation 2 failed.");

    // ask bufferSize2 bytes for external memory
    checkCuSparseError(hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, matA, matB, &beta, matC,
                               computeType, HIPSPARSE_SPGEMM_DEFAULT,
                               spgemmDesc, &bufferSize2, NULL), "hipsparseSpGEMM_compute 1 failed");
    checkCudaError(hipMalloc((void**) &dBuffer2, bufferSize2), "buffer 2 allocation failed");

    // compute A * B
    checkCuSparseError(hipsparseSpGEMM_compute(handle, opA, opB,
                                           &alpha, matA, matB, &beta, matC,
                                           computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDesc, &bufferSize2, dBuffer2), "hipsparseSpGEMM_compute 2 failed");
    // get matrix C sizes
    int64_t rows_C, cols_C, nnzC;
    checkCuSparseError(hipsparseSpMatGetSize(matC, &rows_C, &cols_C, &nnzC), "matC get size failed");
    assert(rows_C == A.rows());
    assert(cols_C == B.cols());

    // Allocate memory for C
    C.rows_ = A.rows();
    C.cols_ = B.cols();
    C.row_offsets = thrust::device_vector<int>(A.rows()+1);
    C.col_ids = thrust::device_vector<int>(nnzC);
    C.data = thrust::device_vector<float>(nnzC);

    // update matC with the new pointers
    checkCuSparseError(hipsparseCsrSetPointers(matC, thrust::raw_pointer_cast(C.row_offsets.data()), 
                                                    thrust::raw_pointer_cast(C.col_ids.data()), 
                                                    thrust::raw_pointer_cast(C.data.data())), "Setting matC pointers failed");

    // copy the final products to the matrix C.
    checkCuSparseError(hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc), "Copying to matC failed");

    // ############################
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    // ############################

    hipEventElapsedTime(&duration, start, stop);

    checkCuSparseError(hipsparseSpGEMM_destroyDescr(spgemmDesc), "SPGEMM descriptor destruction failed");
    checkCuSparseError(hipsparseDestroySpMat(matA), "Matrix descriptor destruction failed");
    checkCuSparseError(hipsparseDestroySpMat(matB), "Matrix descriptor destruction failed");
    checkCuSparseError(hipsparseDestroySpMat(matC), "Matrix descriptor destruction failed");
    checkCudaError(hipFree(dBuffer1), "dBuffer1 free failed");
    checkCudaError(hipFree(dBuffer2), "dBuffer2 free failed");

    return C;
}

thrust::device_vector<float> multiply(hipsparseHandle_t handle, const dCSR& A, const thrust::device_vector<float>& x)
{
    throw std::runtime_error("not implemented yet"); 
    return thrust::device_vector<float>(0);
}

std::tuple<thrust::device_vector<int>, const thrust::device_vector<int>&, const thrust::device_vector<float>&> dCSR::export_coo(hipsparseHandle_t handle) const
{
    thrust::device_vector<int> row_ids(nnz());

    hipsparseXcsr2coo(handle, thrust::raw_pointer_cast(row_offsets.data()), nnz(), cols(), thrust::raw_pointer_cast(row_ids.data()), HIPSPARSE_INDEX_BASE_ZERO); // TODO: should be rows?
            
    return {row_ids, col_ids, data}; 
}

thrust::device_vector<int> dCSR::row_ids(hipsparseHandle_t handle) const
{
    thrust::device_vector<int> _row_ids(nnz());

    hipsparseXcsr2coo(handle, thrust::raw_pointer_cast(row_offsets.data()), nnz(), cols(), thrust::raw_pointer_cast(_row_ids.data()), HIPSPARSE_INDEX_BASE_ZERO);
            
    return _row_ids;
}

struct diag_to_zero_func
{
    __host__ __device__
        void operator()(thrust::tuple<int&,int&,float&> t)
        {
            if(thrust::get<0>(t) == thrust::get<1>(t))
                thrust::get<2>(t) = 0.0;
        }
};
void dCSR::set_diagonal_to_zero(hipsparseHandle_t handle)
{
    thrust::device_vector<int> _row_ids = row_ids(handle);
    
     auto begin = thrust::make_zip_iterator(thrust::make_tuple(col_ids.begin(), _row_ids.begin(), data.begin()));
     auto end = thrust::make_zip_iterator(thrust::make_tuple(col_ids.end(), _row_ids.end(), data.end()));

     thrust::for_each(thrust::device, begin, end, diag_to_zero_func());
}

struct diag_func
{
    float* d;
    __host__ __device__
        void operator()(thrust::tuple<int,int,float> t)
        {
            if(thrust::get<0>(t) == thrust::get<1>(t))
            {
                assert(d[thrust::get<0>(t)] == 0.0);
                d[thrust::get<0>(t)] = thrust::get<2>(t);
            }
        }
};
thrust::device_vector<float> dCSR::diagonal(hipsparseHandle_t handle) const
{
    assert(cols() == rows());
    thrust::device_vector<float> d(rows(), 0.0);

    thrust::device_vector<int> _row_ids = row_ids(handle);

    auto begin = thrust::make_zip_iterator(thrust::make_tuple(col_ids.begin(), _row_ids.begin(), data.begin()));
    auto end = thrust::make_zip_iterator(thrust::make_tuple(col_ids.end(), _row_ids.end(), data.end()));

    thrust::for_each(begin, end, diag_func({thrust::raw_pointer_cast(d.data())})); 

    return d;
}

float dCSR::sum()
{
    return thrust::reduce(data.begin(), data.end(), (float) 0.0, thrust::plus<float>());
}

thrust::device_vector<int> dCSR::compute_cc(const int device)
{
    thrust::device_vector<int> cc_ids(rows());
    computeCC_gpu(rows(), nnz(), 
                thrust::raw_pointer_cast(row_offsets.data()), 
                thrust::raw_pointer_cast(col_ids.data()), 
                thrust::raw_pointer_cast(cc_ids.data()), device);
    return cc_ids;
}

thrust::device_vector<int> dCSR::compute_row_offsets(hipsparseHandle_t handle, const int rows, const thrust::device_vector<int>& col_ids, const thrust::device_vector<int>& row_ids)
{
    assert(row_ids.size() == col_ids.size());
    assert(rows > *thrust::max_element(row_ids.begin(), row_ids.end()));
    assert(thrust::is_sorted(row_ids.begin(), row_ids.end()));
    thrust::device_vector<int> row_offsets(rows+1);
    hipsparseXcoo2csr(handle, thrust::raw_pointer_cast(row_ids.data()), row_ids.size(), rows, thrust::raw_pointer_cast(row_offsets.data()), HIPSPARSE_INDEX_BASE_ZERO);
    return row_offsets;
}

void dCSR::print_info_of(const int i) const
{   
    std::cout<<"Row offsets of "<<i<<", start: "<<row_offsets[i]<<", end excl.: "<<row_offsets[i+1]<<std::endl;
    std::cout<<"Neighbours:"<<std::endl;
    for(size_t l=row_offsets[i]; l<row_offsets[i+1]; ++l)
        std::cout << i << "," << col_ids[l] << "," << data[l] << "\n"; 
}

__global__ void normalize_rows_cuda(const int num_rows, const int* const __restrict__ row_offsets, const int* const __restrict__ col_ids, float* __restrict__ data)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for (int r = tid; r < num_rows; r += num_threads) 
    {
        float sum = 0.0;
        for(int l = row_offsets[r]; l < row_offsets[r + 1]; ++l)
            sum += data[l];

        for(int l = row_offsets[r]; l < row_offsets[r + 1]; ++l)
            data[l] /= sum;

        __syncthreads();
    }
}

void dCSR::normalize_rows()
{
    int threadCount = 256;
    int blockCount = ceil(rows_ / (float) threadCount);

    normalize_rows_cuda<<<blockCount, threadCount>>>(rows_, 
        thrust::raw_pointer_cast(row_offsets.data()), 
        thrust::raw_pointer_cast(col_ids.data()), 
        thrust::raw_pointer_cast(data.data()));
}

std::tuple<thrust::device_vector<unsigned int>, thrust::device_vector<unsigned int>> dCSR::get_spECK_ids()
{
    thrust::device_vector<unsigned int> row_offsets_u(row_offsets.begin(), row_offsets.end());
    thrust::device_vector<unsigned int> col_ids_u(col_ids.begin(), col_ids.end());
    return {row_offsets_u, col_ids_u};
}

dCSR multiply_spECK(hipsparseHandle_t handle, dCSR& A, dCSR& B)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME

    // std::cout<<"\n A_orig \n";
    // A.print();
    thrust::device_vector<unsigned int> row_offsets_u_A, col_ids_u_A, row_offsets_u_B, col_ids_u_B;
    std::tie(row_offsets_u_A, col_ids_u_A) = A.get_spECK_ids();
    std::tie(row_offsets_u_B, col_ids_u_B) = B.get_spECK_ids();

    // spECKWrapper::dCSR<float> A_sp = A.get_spECK_matrix(row_offsets_u_A, col_ids_u_A);
    // spECKWrapper::CSR<float> A_sp_cpu;
    
    // std::cout<<"Writing matrices\n";
    // convert(A_sp_cpu, A_sp, 0);
    // std::string csrPathA = "A_big.hicsr";
    // storeCSR(A_sp_cpu, csrPathA.c_str());
    // std::cout<<"\n A_cpu \n";
    // spECKWrapper::print<float>(A_sp_cpu);

    spECKWrapper::dCSR<float> res_sp;

    auto config = spECK::spECKConfig::initialize(get_cuda_device());

    Timings timings;
    spECK::MultiplyspECK_raw<float, 4, 1024, spECK_DYNAMIC_MEM_PER_BLOCK, spECK_STATIC_MEM_PER_BLOCK>(
        thrust::raw_pointer_cast(row_offsets_u_A.data()), thrust::raw_pointer_cast(col_ids_u_A.data()), thrust::raw_pointer_cast(A.data.data()),
        A.rows(), A.cols(), A.nnz(),
        thrust::raw_pointer_cast(row_offsets_u_B.data()), thrust::raw_pointer_cast(col_ids_u_B.data()), thrust::raw_pointer_cast(B.data.data()),
        B.rows(), B.cols(), B.nnz(),
        res_sp, config, timings);

    dCSR res;
    res.rows_ = res_sp.rows;
    res.cols_ = res_sp.cols;

    thrust::device_ptr<unsigned int> row_offsets_ptr = thrust::device_pointer_cast(res_sp.row_offsets);
    thrust::device_ptr<unsigned int> col_ids_ptr = thrust::device_pointer_cast(res_sp.col_ids);
    thrust::device_ptr<float> data_ptr = thrust::device_pointer_cast(res_sp.data);

    // copy memory to a new device_vector (which automatically allocates memory)
    res.row_offsets = thrust::device_vector<int>(row_offsets_ptr, row_offsets_ptr + res_sp.rows + 1);
    res.col_ids = thrust::device_vector<int>(col_ids_ptr, col_ids_ptr + res_sp.nnz); 
    res.data = thrust::device_vector<float>(data_ptr, data_ptr + res_sp.nnz);

    return res; 
}