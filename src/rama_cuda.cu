#include <hip/hip_runtime.h>
#include "union_find.hxx"
#include "time_measure_util.h"
#include <algorithm>
#include <cstdlib>
#include "ECLgraph.h"
#include <thrust/transform_scan.h>
#include <thrust/transform.h>
#include "maximum_matching_vertex_based.h"
#include "maximum_matching_thrust.h"
#include "multicut_solver_options.h"
#include "dual_solver.h"
#include "edge_contractions_woc.h"
#include "rama_utils.h"

struct is_negative
{
    __host__ __device__
        bool operator()(const float x)
        {
            return x < 0.0;
        }
};
bool has_bad_contractions(const dCOO& A)
{
    const thrust::device_vector<float> d = A.diagonal();
    return thrust::count_if(d.begin(), d.end(), is_negative()) > 0;
}

std::tuple<thrust::device_vector<int>, int> contraction_mapping_by_maximum_matching(dCOO& A, const float mean_multiplier_mm, const bool verbose = true)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;
    thrust::device_vector<int> node_mapping;
    int nr_matched_edges;
    std::tie(node_mapping, nr_matched_edges) = filter_edges_by_matching_vertex_based(A.export_undirected(), mean_multiplier_mm, verbose);
    // std::tie(node_mapping, nr_matched_edges) = filter_edges_by_matching_thrust(A, mean_multiplier_mm, verbose);
    return {compress_label_sequence(node_mapping, node_mapping.size() - 1), nr_matched_edges};
}

std::tuple<thrust::device_vector<int>, double, std::vector<std::vector<int>> > rama_cuda(dCOO& A, const multicut_solver_options& opts)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    assert(A.is_directed());

    const double final_lb = dual_solver(A, opts.max_cycle_length_lb, opts.num_dual_itr_lb, opts.tri_memory_factor, opts.num_outer_itr_dual, 1e-4, opts.verbose);

    if (opts.verbose)
        std::cout << "initial energy = " << A.sum() << "\n";

    thrust::device_vector<int> node_mapping(A.max_dim());
    thrust::sequence(node_mapping.begin(), node_mapping.end());

    std::vector<std::vector<int>> timeline;

    if (opts.only_compute_lb)
        return {std::vector<int>(), final_lb, timeline};
        
    bool try_edges_to_contract_by_maximum_matching = true;
    if (opts.matching_thresh_crossover_ratio > 1.0)
        try_edges_to_contract_by_maximum_matching = false;
    
    for(size_t iter=0; A.nnz() > 0; ++iter)
    {
        if (iter > 0)
        {
            dual_solver(A, opts.max_cycle_length_primal, opts.num_dual_itr_primal, 1.0, 1, 1e-4, opts.verbose);
        }
        thrust::device_vector<int> cur_node_mapping;
        int nr_edges_to_contract;
        if(try_edges_to_contract_by_maximum_matching)
        {
            std::tie(cur_node_mapping, nr_edges_to_contract) = contraction_mapping_by_maximum_matching(A, opts.mean_multiplier_mm, opts.verbose);
            if(nr_edges_to_contract < A.rows() * opts.matching_thresh_crossover_ratio)
            {
                if (opts.verbose)
                {
                    std::cout << "# edges to contract = " << nr_edges_to_contract << ", # vertices = " << A.rows() << "\n";
                    std::cout << "switching to MST based contraction edge selection\n";
                }
                try_edges_to_contract_by_maximum_matching = false;    
            }
        }
        else
        {
            edge_contractions_woc c_mapper(A, opts.verbose);
            std::tie(cur_node_mapping, nr_edges_to_contract) = c_mapper.find_contraction_mapping();
        }

        if(nr_edges_to_contract == 0)
        {
            if (opts.verbose)
                std::cout << "# iterations = " << iter << "\n";
            break;
        }

        dCOO new_A = A.contract_cuda(cur_node_mapping);
        if (opts.verbose)
        {
            std::cout << "original A size " << A.cols() << "x" << A.rows() << "\n";
            std::cout << "contracted A size " << new_A.cols() << "x" << new_A.rows() << "\n";
        }
        assert(new_A.cols() < A.cols());

        if (opts.verbose)
        {
            const thrust::device_vector<float> diagonal = new_A.diagonal();
            const float energy_reduction = thrust::reduce(diagonal.begin(), diagonal.end());
            std::cout << "energy reduction " << energy_reduction << "\n";
        }
        if(has_bad_contractions(new_A))
            throw std::runtime_error("Found bad contractions");

        thrust::swap(A, new_A);
        A.remove_diagonal();
        if (opts.verbose)
            std::cout << "energy after iteration " << iter << ": " << A.sum() << ", #components = " << A.cols() << "\n";
        thrust::gather(node_mapping.begin(), node_mapping.end(), cur_node_mapping.begin(), node_mapping.begin());
        if (opts.dump_timeline)
        {
            std::vector<int> current_timeline(node_mapping.size());
            thrust::copy(node_mapping.begin(), node_mapping.end(), current_timeline.begin());
            timeline.push_back(current_timeline);
        }
        if (opts.max_time_sec >= 0)
        {
            std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
            auto time = std::chrono::duration_cast<std::chrono::seconds>(end - begin).count();
            if (time > opts.max_time_sec)
                break;
        }
    }

    if (opts.verbose)
        std::cout << "final energy = " << A.sum() << "\n";

    return {node_mapping, final_lb, timeline};
}

std::tuple<std::vector<int>, double, int, std::vector<std::vector<int>> > rama_cuda(const std::vector<int>& i, const std::vector<int>& j, const std::vector<float>& costs, const multicut_solver_options& opts, const bool contains_duplicate_edges)
{
    const int cuda_device = get_cuda_device();
    hipSetDevice(cuda_device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device);
    if (opts.verbose)
        std::cout << "Going to use " << prop.name << " " << prop.major << "." << prop.minor << ", device number " << cuda_device << "\n";

    dCOO A(i.begin(), i.end(), j.begin(), j.end(), costs.begin(), costs.end(), true);
    if (contains_duplicate_edges)
    {
        thrust::device_vector<int> identity_mapping(A.max_dim());
        thrust::sequence(identity_mapping.begin(), identity_mapping.end());
        A = A.contract_cuda(identity_mapping); // Merge duplicate edges and sum the costs.
    }
    thrust::device_vector<int> node_mapping;
    double lb;
    std::vector<std::vector<int>> timeline;
    
    std::chrono::steady_clock::time_point start_time = std::chrono::steady_clock::now();
    std::tie(node_mapping, lb, timeline) = rama_cuda(A, opts);
    std::vector<int> h_node_mapping(node_mapping.size());
    thrust::copy(node_mapping.begin(), node_mapping.end(), h_node_mapping.begin());
    std::chrono::steady_clock::time_point end_time = std::chrono::steady_clock::now();
    int time_duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
    return {h_node_mapping, lb, time_duration, timeline};
}

std::tuple<thrust::device_vector<int>, double> rama_cuda(thrust::device_vector<int>&& i, thrust::device_vector<int>&& j, thrust::device_vector<float>&& costs, const multicut_solver_options& opts, const int device)
{
    hipSetDevice(device);
    dCOO A(std::move(j), std::move(i), std::move(costs), true);
    if (contains_duplicate_edges)
    {
        thrust::device_vector<int> identity_mapping(A.max_dim());
        thrust::sequence(identity_mapping.begin(), identity_mapping.end());
        A = A.contract_cuda(identity_mapping); // Merge duplicate edges and sum the costs.
    }
    thrust::device_vector<int> node_mapping;
    double lb;
    std::vector<std::vector<int>> timeline;
    
    std::tie(node_mapping, lb, timeline) = rama_cuda(A, opts);
    return {node_mapping, lb};
}