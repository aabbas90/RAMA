#include "hip/hip_runtime.h"
#include "icp_small_cycles.h"
#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include "time_measure_util.h"
#include <thrust/partition.h>
#include "utils.h"

#define tol 1e-6 

__device__ float get_CSR_value(const int row_index,
                                const int col_id,
                                const int A_num_rows,
                                const int* const __restrict__ row_offsets,
                                const int* const __restrict__ col_ids, 
                                const float* const __restrict__ data, 
                                int& found_index)
{
    if (row_index >= A_num_rows)
        return 0.0;

    for(int l = row_offsets[row_index]; l < row_offsets[row_index + 1]; ++l)
    {
        const int current_col_id = col_ids[l]; 
        // TODO: Binary search
        // TODO: By finding collision between two sorted arrays.
        if (current_col_id > col_id) // col_ids are sorted.
            return 0.0f;

        if (current_col_id == col_id)
        {
            found_index = l;
            return data[l];
        }
    }
    return 0.0f;
}

__device__ float get_CSR_value_both_dir_geq_tol(const int row_index,
                                                const int col_id,
                                                const int A_num_rows,
                                                const int* const __restrict__ row_offsets,
                                                const int* const __restrict__ col_ids, 
                                                const float* const __restrict__ data, 
                                                int& found_index)
{
    float val = get_CSR_value(row_index, col_id, A_num_rows, row_offsets, col_ids, data, found_index);
    if (val < tol) // try other direction.
        val = get_CSR_value(col_id, row_index, A_num_rows, row_offsets, col_ids, data, found_index);
    
    return val;
}

__device__ bool are_connected_by(const int v1, const int v2, const int mid, 
                                const int* const __restrict__ row_offsets, 
                                const int* const __restrict__ col_ids, 
                                const float* const __restrict__ data, const int A_num_rows,
                                int& v1_mid_edge_index, int& v2_mid_edge_index,
                                float& v1_mid_edge_val, float& v2_mid_edge_val)
{
    v1_mid_edge_val = get_CSR_value_both_dir_geq_tol(v1, mid, A_num_rows, row_offsets, col_ids, data, v1_mid_edge_index);
    if (v1_mid_edge_val < tol)
        return false;

    v2_mid_edge_val = get_CSR_value_both_dir_geq_tol(v2, mid, A_num_rows, row_offsets, col_ids, data, v2_mid_edge_index);
    if (v2_mid_edge_val < tol)
        return false;

    return true;
}

// TODO: would be useful if we only want to enumerate triangles and not packing.
// Assumes a symmetric CSR matrix.
// Initialize v1_mid_edge_index by row_offsets[v1] and v2_mid_edge_index by row_offsets[v2].
__device__ int compute_lowest_common_neighbour(const int v1, const int v2, 
                                            const int* const __restrict__ row_offsets, 
                                            const int* const __restrict__ col_ids, 
                                            const float* const __restrict__ data,
                                            int& v1_mid_edge_index, int& v2_mid_edge_index)
{
    while(v1_mid_edge_index < row_offsets[v1 + 1] && v2_mid_edge_index < row_offsets[v2 + 1])
    {
        int v1_n = col_ids[v1_mid_edge_index];
        int v2_n = col_ids[v2_mid_edge_index];
        float v1_mid_edge_value = data[v1_mid_edge_index];
        float v2_mid_edge_value = data[v2_mid_edge_index];
        if (v1_n == v2_n && min(v1_mid_edge_value, v2_mid_edge_value) > tol)
        {
            v1_mid_edge_index++;
            v2_mid_edge_index++;
            return v1_n;
        }
        if (v1_n < v2_n || v1_mid_edge_value < tol)
            ++v1_mid_edge_index;
        if (v1_n > v2_n || v2_mid_edge_value < tol)
            ++v2_mid_edge_index;        
    }
    return -1;
}

__device__ void write_triangle(int* const __restrict__ tri_v1, 
                            int* const __restrict__ tri_v2, 
                            int* const __restrict__ tri_v3, 
                            int* __restrict__ empty_tri_index, 
                            const int v1, const int v2, const int v3)
{
    const int old_index = atomicAdd(empty_tri_index, 1);
    const int min_v = min(v1, min(v2, v3));
    const int max_v = max(v1, max(v2, v3));
    tri_v1[old_index] = min_v;
    tri_v2[old_index] = max(min(v1, v2), min(max(v1, v2), v3));
    tri_v3[old_index] = max_v;
}

__global__ void find_triangles_parallel(const int num_rep_edges,
                                    const int* const __restrict__ row_ids_rep, 
                                    const int* const __restrict__ col_ids_rep, 
                                    const int* const __restrict__ A_symm_row_offsets,
                                    const int* const __restrict__ A_symm_col_ids,
                                    const float* const __restrict__ A_symm_data,
                                    int* __restrict__ triangle_v1,
                                    int* __restrict__ triangle_v2,
                                    int* __restrict__ triangle_v3,
                                    int* __restrict__ empty_tri_index,
                                    const int max_triangles)
{
    const int start_index = blockIdx.x * blockDim.x + threadIdx.x;
    const int num_threads = blockDim.x * gridDim.x;

    for (int edge = start_index; edge < num_rep_edges; edge += num_threads) 
    {
        const int v1 = row_ids_rep[edge];
        const int v2 = col_ids_rep[edge];

        int v1_mid_edge_index = A_symm_row_offsets[v1];
        int v2_mid_edge_index = A_symm_row_offsets[v2];

        while(1)
        {
            int mid = compute_lowest_common_neighbour(v1, v2, 
                                                    A_symm_row_offsets, A_symm_col_ids, A_symm_data,
                                                    v1_mid_edge_index, v2_mid_edge_index);
            if (mid == -1)
                break;

            if (empty_tri_index[0] < max_triangles)
                write_triangle(triangle_v1, triangle_v2, triangle_v3, empty_tri_index, 
                                v1, v2, mid);
        }
        __syncthreads(); //TODO
    }
}

__global__ void pack_quadrangles_parallel(const int num_rep_edges, 
                                        const int* const __restrict__ row_ids_rep, 
                                        const int* const __restrict__ col_ids_rep, 
                                        const int* const __restrict__ A_symm_row_offsets,
                                        const int* const __restrict__ A_symm_col_ids,
                                        const int* const __restrict__ A_row_offsets, // adjacency matrix of original directed graph.
                                        const int* const __restrict__ A_col_ids,
                                        float* __restrict__ A_data,
                                        const int first_rep_edge_index,
                                        const int A_num_rows,
                                        int* __restrict__ triangle_v1,
                                        int* __restrict__ triangle_v2,
                                        int* __restrict__ triangle_v3,
                                        int* __restrict__ empty_tri_index,
                                        const int max_triangles)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int num_threads = blockDim.x * gridDim.x;

    for (int edge = tid + first_rep_edge_index; edge < first_rep_edge_index + num_rep_edges; edge += num_threads) 
    {
        const int v1 = row_ids_rep[edge];
        const int v2 = col_ids_rep[edge];
        int rep_edge_index = -1;
        float rep_edge_cost = get_CSR_value(v1, v2, A_num_rows, A_row_offsets, A_col_ids, A_data, rep_edge_index);
        assert(rep_edge_cost < tol);
        assert(rep_edge_index >= 0); // The repulsive edge must also be present in A.(row -> col).

        // Searching for a path like: v1 -(v1_n1_edge_index)- v1_n1 -(v1_n2_edge_index)- v1_n2 -(v2_edge_index)- v2.
        for(int l1 = A_symm_row_offsets[v1]; l1 < A_symm_row_offsets[v1 + 1] && rep_edge_cost < -tol; ++l1)
        {
            const int v1_n1 = A_symm_col_ids[l1];
            int v1_n1_edge_index, v1_n2_edge_index, v2_edge_index; 
            const float v1_n1_edge_cost = get_CSR_value_both_dir_geq_tol(v1, v1_n1, A_num_rows, A_row_offsets, A_col_ids, A_data, v1_n1_edge_index);
            int v1_n2;
            float v1_n2_edge_cost, v2_edge_cost;
            if (v1_n1_edge_cost > tol)
            {
                for(int l2 = A_symm_row_offsets[v1_n1]; l2 < A_symm_row_offsets[v1_n1 + 1] && rep_edge_cost < -tol; ++l2)
                {
                    v1_n2 = A_symm_col_ids[l2];
                    const bool connected = are_connected_by(v1_n1, v2, v1_n2, 
                                                    A_row_offsets, A_col_ids, A_data, A_num_rows,
                                                    v1_n2_edge_index, v2_edge_index,
                                                    v1_n2_edge_cost, v2_edge_cost);

                    if (connected)
                    {
                        const float packing_value = min(-rep_edge_cost, 
                                                min(v1_n1_edge_cost, 
                                                    min(v1_n2_edge_cost, v2_edge_cost)));

                        rep_edge_cost += packing_value;
                        atomicAdd(&A_data[v1_n1_edge_index], -packing_value);
                        atomicAdd(&A_data[v1_n2_edge_index], -packing_value);
                        atomicAdd(&A_data[v2_edge_index], -packing_value);
                        if (A_data[v1_n1_edge_index] < 0 || A_data[v1_n2_edge_index] < 0 || A_data[v2_edge_index] < 0)
                        {// Undo:
                            rep_edge_cost -= packing_value;
                            atomicAdd(&A_data[v1_n1_edge_index], packing_value);
                            atomicAdd(&A_data[v1_n2_edge_index], packing_value);
                            atomicAdd(&A_data[v2_edge_index], packing_value);    
                        }

                        if (empty_tri_index[0] < max_triangles)
                            write_triangle(triangle_v1, triangle_v2, triangle_v3, empty_tri_index, v1, v1_n1, v2);
                        if (empty_tri_index[0] < max_triangles)
                            write_triangle(triangle_v1, triangle_v2, triangle_v3, empty_tri_index, v1_n1, v1_n2, v2);
                    }
                }
            }
        }
        A_data[rep_edge_index] = rep_edge_cost;
        __syncthreads();
    }
}

__global__ void pack_pentagons_parallel(const int num_rep_edges, 
    const int* const __restrict__ row_ids_rep, 
    const int* const __restrict__ col_ids_rep, 
    const int* const __restrict__ A_symm_row_offsets,
    const int* const __restrict__ A_symm_col_ids,
    const int* const __restrict__ A_row_offsets, // adjacency matrix of original directed graph.
    const int* const __restrict__ A_col_ids,
    float* __restrict__ A_data,
    const int first_rep_edge_index,
    const int A_num_rows)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int num_threads = blockDim.x * gridDim.x;

    for (int edge = tid + first_rep_edge_index; edge < first_rep_edge_index + num_rep_edges; edge += num_threads) 
    {
        const int v1 = row_ids_rep[edge];
        const int v2 = col_ids_rep[edge];
        int rep_edge_index = -1;
        float rep_edge_cost = get_CSR_value(v1, v2, A_num_rows, A_row_offsets, A_col_ids, A_data, rep_edge_index);
        assert(rep_edge_cost < tol);
        assert(rep_edge_index >= 0); // The repulsive edge must also be present in A.(row -> col).

        int v1_n1_edge_index, v1_n2_edge_index, v2_n1_edge_index, v2_edge_index; 
        // Searching for a path like: v1 -(v1_n1_edge_index)- v1_n1 -(v1_n2_edge_index)- v1_n2 -(v2_n1_edge_index)- v2_n1 -(v2_edge_index)-  v2.
        for(int l1 = A_symm_row_offsets[v1]; l1 < A_symm_row_offsets[v1 + 1] && rep_edge_cost < -tol; ++l1)
        {
            const int v1_n1 = A_symm_col_ids[l1];
            const float v1_n1_edge_cost = get_CSR_value_both_dir_geq_tol(v1, v1_n1, A_num_rows, A_row_offsets, A_col_ids, A_data, v1_n1_edge_index);
            if (v1_n1_edge_cost < tol)
                continue; 

            for(int l2 = A_symm_row_offsets[v2]; l2 < A_symm_row_offsets[v2 + 1] && rep_edge_cost < -tol; ++l2)
            {
                const int v2_n1 = A_symm_col_ids[l2];
                const float v2_edge_cost = get_CSR_value_both_dir_geq_tol(v2, v2_n1, A_num_rows, A_row_offsets, A_col_ids, A_data, v2_edge_index);
                if (v2_edge_cost < tol)
                    continue;

                for(int l3 = A_symm_row_offsets[v1_n1]; l3 < A_symm_row_offsets[v1_n1 + 1] && rep_edge_cost < -tol; ++l3)
                {
                    const int v1_n2 = A_symm_col_ids[l3];
                    float v1_n2_edge_cost, v2_n1_edge_cost;
                    const bool connected = are_connected_by(v1_n1, v2_n1, v1_n2, 
                                                    A_row_offsets, A_col_ids, A_data, A_num_rows,
                                                    v1_n2_edge_index, v2_n1_edge_index,
                                                    v1_n2_edge_cost, v2_n1_edge_cost);
                    
                    if (!connected)
                        continue;

                    const float packing_value = min(-rep_edge_cost, 
                                            min(v1_n1_edge_cost, 
                                                min(v1_n2_edge_cost,
                                                    min(v2_n1_edge_cost, v2_edge_cost))));

                    rep_edge_cost += packing_value;
                    atomicAdd(&A_data[v1_n1_edge_index], -packing_value);
                    atomicAdd(&A_data[v1_n2_edge_index], -packing_value);
                    atomicAdd(&A_data[v2_n1_edge_index], -packing_value);
                    atomicAdd(&A_data[v2_edge_index], -packing_value);
                    if (A_data[v1_n1_edge_index] < 0 || A_data[v1_n2_edge_index] < 0 || A_data[v2_edge_index] < 0 || A_data[v2_n1_edge_index] < 0)
                    {// Undo:
                        rep_edge_cost -= packing_value;
                        atomicAdd(&A_data[v1_n1_edge_index], packing_value);
                        atomicAdd(&A_data[v1_n2_edge_index], packing_value);
                        atomicAdd(&A_data[v2_n1_edge_index], packing_value);
                        atomicAdd(&A_data[v2_edge_index], packing_value);    
                    }
                }
            }
        }
        A_data[rep_edge_index] = rep_edge_cost;
        __syncthreads();
    }
}

struct is_positive_edge
{
    __host__ __device__ bool operator()(const thrust::tuple<int,int,float>& t)
    {
        if(thrust::get<2>(t) > 0.0f)
            return true;
        else
            return false;
    }
};

std::tuple<dCOO, thrust::device_vector<int>, thrust::device_vector<int>, int> create_matrices(hipsparseHandle_t handle, const dCOO& A)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
    
    // Partition edges into positive and negative.
    thrust::device_vector<int> row_ids_rep = A.get_row_ids();
    thrust::device_vector<int> col_ids_rep = A.get_col_ids();
    thrust::device_vector<float> costs = A.get_data();

    auto first = thrust::make_zip_iterator(thrust::make_tuple(row_ids_rep.begin(), col_ids_rep.begin(), costs.begin()));
    auto last = thrust::make_zip_iterator(thrust::make_tuple(row_ids_rep.end(), col_ids_rep.end(), costs.end()));

    auto first_negative = thrust::partition(first, last, is_positive_edge());
    const size_t nr_positive_edges = std::distance(first, first_negative);

    // Create symmetric adjacency matrix of positive edges.
    thrust::device_vector<int> pos_row_ids_symm, pos_col_ids_symm;
    thrust::device_vector<float> pos_costs_symm;
    dCOO A_pos;
    if (nr_positive_edges > 0)
    {
        std::tie(pos_row_ids_symm, pos_col_ids_symm, pos_costs_symm) = to_undirected(row_ids_rep.begin(), row_ids_rep.begin() + nr_positive_edges,
                                                                                    col_ids_rep.begin(), col_ids_rep.begin() + nr_positive_edges,
                                                                                    costs.begin(), costs.begin() + nr_positive_edges);
        A_pos = dCOO(std::max(A.rows(), A.cols()), std::max(A.rows(), A.cols()),
                    std::move(pos_col_ids_symm), 
                    std::move(pos_row_ids_symm), 
                    std::move(pos_costs_symm));
        row_ids_rep = thrust::device_vector<int>(row_ids_rep.begin() + nr_positive_edges, row_ids_rep.end());
        col_ids_rep = thrust::device_vector<int>(col_ids_rep.begin() + nr_positive_edges, col_ids_rep.end());
    }
    return {A_pos, row_ids_rep, col_ids_rep, nr_positive_edges};
}

// A should be directed thus containing same number of elements as in original problem. Does packing in-place on A.
std::tuple<double, thrust::device_vector<int>, thrust::device_vector<int>, thrust::device_vector<int>> parallel_small_cycle_packing_cuda(hipsparseHandle_t handle, dCOO& A, const int max_tries_triangles, const int max_tries_quads)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;

    int num_nodes = A.rows();
    double lb = get_lb(A.get_data());
    std::cout<<"Initial lb: "<<lb<<std::endl;

    // Make adjacency matrix and BFS search starting matrix.
    dCOO A_pos;
    thrust::device_vector<int> row_ids_rep, col_ids_rep;
    int nr_positive_edges;
    std::tie(A_pos, row_ids_rep, col_ids_rep, nr_positive_edges) = create_matrices(handle, A);
    if (nr_positive_edges == 0)
        return {lb, thrust::device_vector<int>(0), thrust::device_vector<int>(0), thrust::device_vector<int>(0)}; 

    int num_rep_edges = row_ids_rep.size();
 
    thrust::device_vector<int> A_pos_row_offsets = A_pos.compute_row_offsets(handle);

    int threadCount = 256;
    int blockCount = ceil(num_rep_edges / (float) threadCount);
    thrust::device_vector<int> triangles_v1(num_rep_edges * 10); //TODO
    thrust::device_vector<int> triangles_v2(num_rep_edges * 10); //TODO
    thrust::device_vector<int> triangles_v3(num_rep_edges * 10); //TODO
    thrust::device_vector<int> empty_tri_index(1, 0);

    pack_triangles_parallel<<<blockCount, threadCount>>>(num_rep_edges, 
        thrust::raw_pointer_cast(row_ids_rep.data()), 
        thrust::raw_pointer_cast(col_ids_rep.data()), 
        thrust::raw_pointer_cast(A_pos_row_offsets.data()),
        A_pos.get_col_ids_ptr(),
        A_pos.get_data_ptr(),
        thrust::raw_pointer_cast(triangles_v1.data()),
        thrust::raw_pointer_cast(triangles_v2.data()),
        thrust::raw_pointer_cast(triangles_v3.data()),
        thrust::raw_pointer_cast(empty_tri_index.data()),
        triangles_v1.size());
    
    std::cout<<"3-cycles: found # of triangles: "<<empty_tri_index[0]<<", budget: "<<triangles_v1.size()<<std::endl;
    int nr_triangles = empty_tri_index[0];
    triangles_v1.resize(nr_triangles);
    triangles_v2.resize(nr_triangles);
    triangles_v3.resize(nr_triangles);
    return {lb, triangles_v1, triangles_v2, triangles_v3};
}

std::tuple<double, dCOO, thrust::device_vector<int>, thrust::device_vector<int>, thrust::device_vector<int>> parallel_small_cycle_packing_cuda(const std::vector<int>& i, const std::vector<int>& j, const std::vector<float>& costs, const int max_tries_triangles, const int max_tries_quads)
{
    const int cuda_device = get_cuda_device();
    hipSetDevice(cuda_device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device);
    std::cout << "Going to use " << prop.name << " " << prop.major << "." << prop.minor << ", device number " << cuda_device << "\n";
    hipsparseHandle_t handle;
    checkCuSparseError(hipsparseCreate(&handle), "cusparse init failed");
    
    dCOO A(i.begin(), i.end(),
        j.begin(), j.end(), 
        costs.begin(), costs.end());
    
    thrust::device_vector<int> triangles_v1, triangles_v2, triangles_v3;
    double lb;
    std::tie(lb, triangles_v1, triangles_v2, triangles_v3) = parallel_small_cycle_packing_cuda(handle, A, max_tries_triangles, max_tries_quads);
    return {lb, A, triangles_v1, triangles_v2, triangles_v3};
}

double parallel_small_cycle_packing_cuda_lower_bound(const std::vector<int>& i, const std::vector<int>& j, const std::vector<float>& costs, const int max_tries_triangles, const int max_tries_quads)
{
    dCOO A; 
    double lb;
    thrust::device_vector<int> triangles_v1, triangles_v2, triangles_v3;
    std::tie(lb, A, triangles_v1, triangles_v2, triangles_v3) = parallel_small_cycle_packing_cuda(i, j, costs, max_tries_triangles, max_tries_quads);
    return lb;
}