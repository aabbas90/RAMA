#include "hip/hip_runtime.h"
#include "icp_small_cycles.h"
#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include "time_measure_util.h"
#include <thrust/partition.h>
#include "utils.h"

#define tol 1e-6 

__device__ float get_CSR_value(const int row_index,
                                const int col_id,
                                const int A_num_rows,
                                const int* const __restrict__ row_offsets,
                                const int* const __restrict__ col_ids, 
                                const float* const __restrict__ data, 
                                int& found_index)
{
    if (row_index >= A_num_rows)
        return 0.0;

    for(int l = row_offsets[row_index]; l < row_offsets[row_index + 1]; ++l)
    {
        const int current_col_id = col_ids[l]; 
        // TODO: Binary search
        // TODO: By finding collision between two sorted arrays.
        if (current_col_id > col_id) // col_ids are sorted.
            return 0.0f;

        if (current_col_id == col_id)
        {
            found_index = l;
            return data[l];
        }
    }
    return 0.0f;
}

__device__ float get_CSR_value_both_dir_geq_tol(const int row_index,
                                                const int col_id,
                                                const int A_num_rows,
                                                const int* const __restrict__ row_offsets,
                                                const int* const __restrict__ col_ids, 
                                                const float* const __restrict__ data, 
                                                int& found_index)
{
    float val = get_CSR_value(row_index, col_id, A_num_rows, row_offsets, col_ids, data, found_index);
    if (val < tol) // try other direction.
        val = get_CSR_value(col_id, row_index, A_num_rows, row_offsets, col_ids, data, found_index);
    
    return val;
}

__device__ bool are_connected_by(const int v1, const int v2, const int mid, 
                                const int* const __restrict__ row_offsets, 
                                const int* const __restrict__ col_ids, 
                                const float* const __restrict__ data, const int A_num_rows,
                                int& v1_mid_edge_index, int& v2_mid_edge_index,
                                float& v1_mid_edge_val, float& v2_mid_edge_val)
{
    v1_mid_edge_val = get_CSR_value_both_dir_geq_tol(v1, mid, A_num_rows, row_offsets, col_ids, data, v1_mid_edge_index);
    if (v1_mid_edge_val < tol)
        return false;

    v2_mid_edge_val = get_CSR_value_both_dir_geq_tol(v2, mid, A_num_rows, row_offsets, col_ids, data, v2_mid_edge_index);
    if (v2_mid_edge_val < tol)
        return false;

    return true;
}

// Assumes a symmetric CSR matrix.
// Initialize v1_mid_edge_index by row_offsets[v1] and v2_mid_edge_index by row_offsets[v2].
__device__ int compute_lowest_common_neighbour(const int v1, const int v2, 
                                            const int* const __restrict__ row_offsets, 
                                            const int* const __restrict__ col_ids, 
                                            const float* const __restrict__ data,
                                            int& v1_mid_edge_index, int& v2_mid_edge_index)
{
    while(v1_mid_edge_index < row_offsets[v1 + 1] && v2_mid_edge_index < row_offsets[v2 + 1])
    {
        int v1_n = col_ids[v1_mid_edge_index];
        int v2_n = col_ids[v2_mid_edge_index];
        float v1_mid_edge_value = data[v1_mid_edge_index];
        float v2_mid_edge_value = data[v2_mid_edge_index];
        if (v1_n == v2_n && min(v1_mid_edge_value, v2_mid_edge_value) > tol)
        {
            v1_mid_edge_index++;
            v2_mid_edge_index++;
            return v1_n;
        }
        if (v1_n < v2_n || v1_mid_edge_value < tol)
            ++v1_mid_edge_index;
        if (v1_n > v2_n || v2_mid_edge_value < tol)
            ++v2_mid_edge_index;        
    }
    return -1;
}

__device__ void write_triangle(int* const __restrict__ tri_v1, 
                            int* const __restrict__ tri_v2, 
                            int* const __restrict__ tri_v3, 
                            int* __restrict__ empty_tri_index, 
                            const int v1, const int v2, const int v3)
{
    const int old_index = atomicAdd(empty_tri_index, 1);
    const int min_v = min(v1, min(v2, v3));
    const int max_v = max(v1, max(v2, v3));
    tri_v1[old_index] = min_v;
    tri_v2[old_index] = max(min(v1, v2), min(max(v1, v2), v3));
    tri_v3[old_index] = max_v;
}

__global__ void find_triangles_parallel(const int num_rep_edges,
                                    const int* const __restrict__ row_ids_rep, 
                                    const int* const __restrict__ col_ids_rep, 
                                    const int* const __restrict__ A_symm_row_offsets,
                                    const int* const __restrict__ A_symm_col_ids,
                                    const float* const __restrict__ A_symm_data,
                                    int* __restrict__ triangle_v1,
                                    int* __restrict__ triangle_v2,
                                    int* __restrict__ triangle_v3,
                                    int* __restrict__ empty_tri_index,
                                    const int max_triangles)
{
    const int start_index = blockIdx.x * blockDim.x + threadIdx.x;
    const int num_threads = blockDim.x * gridDim.x;

    for (int edge = start_index; edge < num_rep_edges; edge += num_threads) 
    {
        const int v1 = row_ids_rep[edge];
        const int v2 = col_ids_rep[edge];

        int v1_mid_edge_index = A_symm_row_offsets[v1];
        int v2_mid_edge_index = A_symm_row_offsets[v2];

        while(1)
        {
            int mid = compute_lowest_common_neighbour(v1, v2, 
                                                    A_symm_row_offsets, A_symm_col_ids, A_symm_data,
                                                    v1_mid_edge_index, v2_mid_edge_index);
            if (mid == -1)
                break;

            if (empty_tri_index[0] < max_triangles)
                write_triangle(triangle_v1, triangle_v2, triangle_v3, empty_tri_index, 
                                v1, v2, mid);
        }
        __syncthreads(); //TODO
    }
}

__global__ void pack_quadrangles_parallel(const int num_rep_edges, 
                                        const int* const __restrict__ row_ids_rep, 
                                        const int* const __restrict__ col_ids_rep, 
                                        const int* const __restrict__ A_symm_row_offsets,
                                        const int* const __restrict__ A_symm_col_ids,
                                        const int* const __restrict__ A_row_offsets, // adjacency matrix of original directed graph.
                                        const int* const __restrict__ A_col_ids,
                                        float* __restrict__ A_data,
                                        const int first_rep_edge_index,
                                        const int A_num_rows,
                                        int* __restrict__ triangle_v1,
                                        int* __restrict__ triangle_v2,
                                        int* __restrict__ triangle_v3,
                                        int* __restrict__ empty_tri_index,
                                        const int max_triangles)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int num_threads = blockDim.x * gridDim.x;

    for (int edge = tid + first_rep_edge_index; edge < first_rep_edge_index + num_rep_edges; edge += num_threads) 
    {
        const int v1 = row_ids_rep[edge];
        const int v2 = col_ids_rep[edge];
        int rep_edge_index = -1;
        float rep_edge_cost = get_CSR_value(v1, v2, A_num_rows, A_row_offsets, A_col_ids, A_data, rep_edge_index);
        assert(rep_edge_cost < tol);
        assert(rep_edge_index >= 0); // The repulsive edge must also be present in A.(row -> col).

        // Searching for a path like: v1 -(v1_n1_edge_index)- v1_n1 -(v1_n2_edge_index)- v1_n2 -(v2_edge_index)- v2.
        for(int l1 = A_symm_row_offsets[v1]; l1 < A_symm_row_offsets[v1 + 1] && rep_edge_cost < -tol; ++l1)
        {
            const int v1_n1 = A_symm_col_ids[l1];
            int v1_n1_edge_index, v1_n2_edge_index, v2_edge_index; 
            const float v1_n1_edge_cost = get_CSR_value_both_dir_geq_tol(v1, v1_n1, A_num_rows, A_row_offsets, A_col_ids, A_data, v1_n1_edge_index);
            int v1_n2;
            float v1_n2_edge_cost, v2_edge_cost;
            if (v1_n1_edge_cost > tol)
            {
                for(int l2 = A_symm_row_offsets[v1_n1]; l2 < A_symm_row_offsets[v1_n1 + 1] && rep_edge_cost < -tol; ++l2)
                {
                    v1_n2 = A_symm_col_ids[l2];
                    const bool connected = are_connected_by(v1_n1, v2, v1_n2, 
                                                    A_row_offsets, A_col_ids, A_data, A_num_rows,
                                                    v1_n2_edge_index, v2_edge_index,
                                                    v1_n2_edge_cost, v2_edge_cost);

                    if (connected)
                    {
                        const float packing_value = min(-rep_edge_cost, 
                                                min(v1_n1_edge_cost, 
                                                    min(v1_n2_edge_cost, v2_edge_cost)));

                        rep_edge_cost += packing_value;
                        atomicAdd(&A_data[v1_n1_edge_index], -packing_value);
                        atomicAdd(&A_data[v1_n2_edge_index], -packing_value);
                        atomicAdd(&A_data[v2_edge_index], -packing_value);
                        if (A_data[v1_n1_edge_index] < 0 || A_data[v1_n2_edge_index] < 0 || A_data[v2_edge_index] < 0)
                        {// Undo:
                            rep_edge_cost -= packing_value;
                            atomicAdd(&A_data[v1_n1_edge_index], packing_value);
                            atomicAdd(&A_data[v1_n2_edge_index], packing_value);
                            atomicAdd(&A_data[v2_edge_index], packing_value);    
                        }

                        if (empty_tri_index[0] < max_triangles)
                            write_triangle(triangle_v1, triangle_v2, triangle_v3, empty_tri_index, v1, v1_n1, v2);
                        if (empty_tri_index[0] < max_triangles)
                            write_triangle(triangle_v1, triangle_v2, triangle_v3, empty_tri_index, v1_n1, v1_n2, v2);
                    }
                }
            }
        }
        A_data[rep_edge_index] = rep_edge_cost;
        __syncthreads();
    }
}

__global__ void pack_pentagons_parallel(const int num_rep_edges, 
    const int* const __restrict__ row_ids_rep, 
    const int* const __restrict__ col_ids_rep, 
    const int* const __restrict__ A_symm_row_offsets,
    const int* const __restrict__ A_symm_col_ids,
    const int* const __restrict__ A_row_offsets, // adjacency matrix of original directed graph.
    const int* const __restrict__ A_col_ids,
    float* __restrict__ A_data,
    const int first_rep_edge_index,
    const int A_num_rows)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int num_threads = blockDim.x * gridDim.x;

    for (int edge = tid + first_rep_edge_index; edge < first_rep_edge_index + num_rep_edges; edge += num_threads) 
    {
        const int v1 = row_ids_rep[edge];
        const int v2 = col_ids_rep[edge];
        int rep_edge_index = -1;
        float rep_edge_cost = get_CSR_value(v1, v2, A_num_rows, A_row_offsets, A_col_ids, A_data, rep_edge_index);
        assert(rep_edge_cost < tol);
        assert(rep_edge_index >= 0); // The repulsive edge must also be present in A.(row -> col).

        int v1_n1_edge_index, v1_n2_edge_index, v2_n1_edge_index, v2_edge_index; 
        // Searching for a path like: v1 -(v1_n1_edge_index)- v1_n1 -(v1_n2_edge_index)- v1_n2 -(v2_n1_edge_index)- v2_n1 -(v2_edge_index)-  v2.
        for(int l1 = A_symm_row_offsets[v1]; l1 < A_symm_row_offsets[v1 + 1] && rep_edge_cost < -tol; ++l1)
        {
            const int v1_n1 = A_symm_col_ids[l1];
            const float v1_n1_edge_cost = get_CSR_value_both_dir_geq_tol(v1, v1_n1, A_num_rows, A_row_offsets, A_col_ids, A_data, v1_n1_edge_index);
            if (v1_n1_edge_cost < tol)
                continue; 

            for(int l2 = A_symm_row_offsets[v2]; l2 < A_symm_row_offsets[v2 + 1] && rep_edge_cost < -tol; ++l2)
            {
                const int v2_n1 = A_symm_col_ids[l2];
                const float v2_edge_cost = get_CSR_value_both_dir_geq_tol(v2, v2_n1, A_num_rows, A_row_offsets, A_col_ids, A_data, v2_edge_index);
                if (v2_edge_cost < tol)
                    continue;

                for(int l3 = A_symm_row_offsets[v1_n1]; l3 < A_symm_row_offsets[v1_n1 + 1] && rep_edge_cost < -tol; ++l3)
                {
                    const int v1_n2 = A_symm_col_ids[l3];
                    float v1_n2_edge_cost, v2_n1_edge_cost;
                    const bool connected = are_connected_by(v1_n1, v2_n1, v1_n2, 
                                                    A_row_offsets, A_col_ids, A_data, A_num_rows,
                                                    v1_n2_edge_index, v2_n1_edge_index,
                                                    v1_n2_edge_cost, v2_n1_edge_cost);
                    
                    if (!connected)
                        continue;

                    const float packing_value = min(-rep_edge_cost, 
                                            min(v1_n1_edge_cost, 
                                                min(v1_n2_edge_cost,
                                                    min(v2_n1_edge_cost, v2_edge_cost))));

                    rep_edge_cost += packing_value;
                    atomicAdd(&A_data[v1_n1_edge_index], -packing_value);
                    atomicAdd(&A_data[v1_n2_edge_index], -packing_value);
                    atomicAdd(&A_data[v2_n1_edge_index], -packing_value);
                    atomicAdd(&A_data[v2_edge_index], -packing_value);
                    if (A_data[v1_n1_edge_index] < 0 || A_data[v1_n2_edge_index] < 0 || A_data[v2_edge_index] < 0 || A_data[v2_n1_edge_index] < 0)
                    {// Undo:
                        rep_edge_cost -= packing_value;
                        atomicAdd(&A_data[v1_n1_edge_index], packing_value);
                        atomicAdd(&A_data[v1_n2_edge_index], packing_value);
                        atomicAdd(&A_data[v2_n1_edge_index], packing_value);
                        atomicAdd(&A_data[v2_edge_index], packing_value);    
                    }
                }
            }
        }
        A_data[rep_edge_index] = rep_edge_cost;
        __syncthreads();
    }
}

struct is_positive_edge
{
    __host__ __device__ bool operator()(const thrust::tuple<int,int,float>& t)
    {
        return thrust::get<2>(t) > tol;
    }
};

struct is_neg_edge
{
    __host__ __device__ bool operator()(const thrust::tuple<int,int,float>& t)
    {
        return thrust::get<2>(t) < tol;
    }
};

std::tuple<dCOO, thrust::device_vector<int>, thrust::device_vector<int>, int> create_matrices(const dCOO& A)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
    
    // Partition edges into positive and negative.
    thrust::device_vector<int> row_ids = A.get_row_ids();
    thrust::device_vector<int> col_ids = A.get_col_ids();
    thrust::device_vector<float> costs = A.get_data();

    auto first = thrust::make_zip_iterator(thrust::make_tuple(row_ids.begin(), col_ids.begin(), costs.begin()));
    auto last = thrust::make_zip_iterator(thrust::make_tuple(row_ids.end(), col_ids.end(), costs.end()));

    thrust::device_vector<int> row_ids_pos(row_ids.size());
    thrust::device_vector<int> col_ids_pos(row_ids.size());
    thrust::device_vector<float> costs_pos(row_ids.size());
    auto first_pos = thrust::make_zip_iterator(thrust::make_tuple(row_ids_pos.begin(), col_ids_pos.begin(), costs_pos.begin()));
    auto last_pos = thrust::copy_if(first, last, first_pos, is_positive_edge());
    const int num_positive_edges = std::distance(first_pos, last_pos);
    row_ids_pos.resize(num_positive_edges);
    col_ids_pos.resize(num_positive_edges);
    costs_pos.resize(num_positive_edges);

    thrust::device_vector<int> row_ids_neg(row_ids.size());
    thrust::device_vector<int> col_ids_neg(row_ids.size());
    thrust::device_vector<float> costs_neg(row_ids.size());
    auto first_neg = thrust::make_zip_iterator(thrust::make_tuple(row_ids_neg.begin(), col_ids_neg.begin(), costs_neg.begin()));
    auto last_neg = thrust::copy_if(first, last, first_neg, is_neg_edge());
    const int nr_neg_edges = std::distance(first_neg, last_neg);
    row_ids_neg.resize(nr_neg_edges);
    col_ids_neg.resize(nr_neg_edges);

    // Create symmetric adjacency matrix of positive edges.
    dCOO A_pos_symm;
    if (num_positive_edges > 0)
    {
        std::tie(row_ids_pos, col_ids_pos, costs_pos) = to_undirected(row_ids_pos, col_ids_pos, costs_pos);
        A_pos_symm = dCOO(std::max(A.rows(), A.cols()), std::max(A.rows(), A.cols()),
                    std::move(col_ids_pos), 
                    std::move(row_ids_pos), 
                    std::move(costs_pos));
    }
    return {A_pos_symm, row_ids_neg, col_ids_neg, num_positive_edges};
}

// A should be directed thus containing same number of elements as in original problem. Does packing in-place on A.
std::tuple<thrust::device_vector<int>, thrust::device_vector<int>, thrust::device_vector<int>> parallel_small_cycle_packing_cuda(hipsparseHandle_t handle, const dCOO& A)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;

    int num_nodes = A.rows();

    // Make adjacency matrix and BFS search starting matrix.
    dCOO A_pos;
    thrust::device_vector<int> row_ids_rep, col_ids_rep;
    int num_positive_edges;
    std::tie(A_pos, row_ids_rep, col_ids_rep, num_positive_edges) = create_matrices(A);
    int num_rep_edges = row_ids_rep.size();
    if (num_positive_edges == 0 || num_rep_edges == 0)
        return {thrust::device_vector<int>(0), thrust::device_vector<int>(0), thrust::device_vector<int>(0)};

    thrust::device_vector<int> A_pos_row_offsets = A_pos.compute_row_offsets(handle);

    int threadCount = 256;
    int blockCount = ceil(num_rep_edges / (float) threadCount);
    thrust::device_vector<int> triangles_v1(num_rep_edges * 10); //TODO
    thrust::device_vector<int> triangles_v2(num_rep_edges * 10); //TODO 
    thrust::device_vector<int> triangles_v3(num_rep_edges * 10); //TODO
    thrust::device_vector<int> empty_tri_index(1, 0);

    find_triangles_parallel<<<blockCount, threadCount>>>(num_rep_edges, 
        thrust::raw_pointer_cast(row_ids_rep.data()), 
        thrust::raw_pointer_cast(col_ids_rep.data()), 
        thrust::raw_pointer_cast(A_pos_row_offsets.data()),
        A_pos.get_col_ids_ptr(),
        A_pos.get_data_ptr(),
        thrust::raw_pointer_cast(triangles_v1.data()),
        thrust::raw_pointer_cast(triangles_v2.data()),
        thrust::raw_pointer_cast(triangles_v3.data()),
        thrust::raw_pointer_cast(empty_tri_index.data()),
        triangles_v1.size());
    
    std::cout<<"3-cycles: found # of triangles: "<<empty_tri_index[0]<<", budget: "<<triangles_v1.size()<<std::endl;
    int nr_triangles = empty_tri_index[0];
    triangles_v1.resize(nr_triangles);
    triangles_v2.resize(nr_triangles);
    triangles_v3.resize(nr_triangles);
    return {triangles_v1, triangles_v2, triangles_v3};
}