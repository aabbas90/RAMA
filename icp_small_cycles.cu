#include "hip/hip_runtime.h"
#include "icp_small_cycles.h"
#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include "time_measure_util.h"
#include <thrust/partition.h>
#include "utils.h"

#define tol 1e-6 

__device__ float get_CSR_value(const int row_index,
                                const int col_id,
                                const int A_num_rows,
                                const int* const __restrict__ row_offsets,
                                const int* const __restrict__ col_ids, 
                                const float* const __restrict__ data, 
                                int& found_index)
{
    if (row_index >= A_num_rows)
        return 0.0;

    for(int l = row_offsets[row_index]; l < row_offsets[row_index + 1]; ++l)
    {
        int current_col_id = col_ids[l]; 
        // TODO: Binary search
        // TODO: By finding collision between two sorted arrays.
        if (current_col_id > col_id) // col_ids are sorted.
            return 0.0f;

        if (current_col_id == col_id)
        {
            found_index = l;
            return data[l];
        }
    }
    return 0.0f;
}

__device__ float get_CSR_value_both_dir_geq_tol(const int row_index,
                                                const int col_id,
                                                const int A_num_rows,
                                                const int* const __restrict__ row_offsets,
                                                const int* const __restrict__ col_ids, 
                                                const float* const __restrict__ data, 
                                                int& found_index)
{
    float val = get_CSR_value(row_index, col_id, A_num_rows, row_offsets, col_ids, data, found_index);
    if (val < tol) // try other direction.
        val = get_CSR_value(col_id, row_index, A_num_rows, row_offsets, col_ids, data, found_index);
    
    return val;
}

__device__ bool are_connected_by(const int v1, const int v2, const int mid, 
                                const int* const __restrict__ row_offsets, 
                                const int* const __restrict__ col_ids, 
                                const float* const __restrict__ data, const int A_num_rows,
                                int& v1_mid_edge_index, int& v2_mid_edge_index,
                                float& v1_mid_edge_val, float& v2_mid_edge_val)
{
    v1_mid_edge_val = get_CSR_value_both_dir_geq_tol(v1, mid, A_num_rows, row_offsets, col_ids, data, v1_mid_edge_index);
    if (v1_mid_edge_val < tol)
        return false;

    v2_mid_edge_val = get_CSR_value_both_dir_geq_tol(v2, mid, A_num_rows, row_offsets, col_ids, data, v2_mid_edge_index);
    if (v2_mid_edge_val < tol)
        return false;

    return true;
}

__global__ void pack_triangles_parallel(const int num_rep_edges,
                                    const int* const __restrict__ row_ids_rep, 
                                    const int* const __restrict__ col_ids_rep, 
                                    const int* const __restrict__ A_symm_row_offsets,
                                    const int* const __restrict__ A_symm_col_ids,
                                    const int* const __restrict__ A_row_offsets, // adjacency matrix of original directed graph.
                                    const int* const __restrict__ A_col_ids,
                                    float* __restrict__ A_data,
                                    const int first_rep_edge_index,
                                    const int A_num_rows,
                                    int3* __restrict__ triangle_vertices,
                                    int* __restrict__ empty_tri_index,
                                    int max_triangles)
{
    int start_index = blockIdx.x * blockDim.x + threadIdx.x + first_rep_edge_index;
    int num_threads = blockDim.x * gridDim.x;

    for (int edge = start_index; edge < first_rep_edge_index + num_rep_edges; edge += num_threads) 
    {
        int rep_edge_row = row_ids_rep[edge];
        int rep_edge_col = col_ids_rep[edge];
        int rep_edge_index = -1;
        float rep_edge_cost = get_CSR_value(rep_edge_row, rep_edge_col, A_num_rows, A_row_offsets, A_col_ids, A_data, rep_edge_index);
        assert(rep_edge_cost < tol);
        assert(rep_edge_index >= 0); // The repulsive edge must also be present in A.(row -> col).
        
        for(int l = A_symm_row_offsets[rep_edge_row]; l < A_symm_row_offsets[rep_edge_row + 1] && rep_edge_cost < -tol; ++l)
        {
            int current_col_id = A_symm_col_ids[l];
            int found_upper_index, found_lower_index;
            float upper_cost, lower_cost;
            bool connected = are_connected_by(rep_edge_row, rep_edge_col, current_col_id, 
                                            A_row_offsets, A_col_ids, A_data, A_num_rows,
                                            found_upper_index, found_lower_index,
                                            upper_cost, lower_cost);

            if (connected)
            {
                float packing_value = min(-rep_edge_cost, min(lower_cost, upper_cost));
                rep_edge_cost += packing_value;
                atomicAdd(&A_data[found_upper_index], -packing_value);
                atomicAdd(&A_data[found_lower_index], -packing_value);
                if (A_data[found_upper_index] < 0 || A_data[found_lower_index] < 0)
                {   // Undo:
                    rep_edge_cost -= packing_value;
                    atomicAdd(&A_data[found_upper_index], packing_value);
                    atomicAdd(&A_data[found_lower_index], packing_value);
                }
                int3 tri = make_int3(rep_edge_row, current_col_id, rep_edge_col);
                if (empty_tri_index[0] < max_triangles)
                    triangle_vertices[atomicAdd(empty_tri_index, 1)] = tri;
            }
        }
        A_data[rep_edge_index] = rep_edge_cost;
        __syncthreads();
    }
}

__global__ void pack_quadrangles_parallel(const int num_rep_edges, 
    const int* const __restrict__ row_ids_rep, 
    const int* const __restrict__ col_ids_rep, 
    const int* const __restrict__ A_symm_row_offsets,
    const int* const __restrict__ A_symm_col_ids,
    const int* const __restrict__ A_row_offsets, // adjacency matrix of original directed graph.
    const int* const __restrict__ A_col_ids,
    float* __restrict__ A_data,
    const int first_rep_edge_index,
    const int A_num_rows,
    int3* __restrict__ triangle_vertices,
    int* __restrict__ empty_tri_index,
    int max_triangles)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for (int edge = tid + first_rep_edge_index; edge < first_rep_edge_index + num_rep_edges; edge += num_threads) 
    {
        int v1 = row_ids_rep[edge];
        int v2 = col_ids_rep[edge];
        int rep_edge_index = -1;
        float rep_edge_cost = get_CSR_value(v1, v2, A_num_rows, A_row_offsets, A_col_ids, A_data, rep_edge_index);
        assert(rep_edge_cost < tol);
        assert(rep_edge_index >= 0); // The repulsive edge must also be present in A.(row -> col).

        // Searching for a path like: v1 -(v1_n1_edge_index)- v1_n1 -(v1_n2_edge_index)- v1_n2 -(v2_edge_index)- v2.
        for(int l1 = A_symm_row_offsets[v1]; l1 < A_symm_row_offsets[v1 + 1] && rep_edge_cost < -tol; ++l1)
        {
            int v1_n1 = A_symm_col_ids[l1];
            int v1_n1_edge_index, v1_n2_edge_index, v2_edge_index; 
            float v1_n1_edge_cost = get_CSR_value_both_dir_geq_tol(v1, v1_n1, A_num_rows, A_row_offsets, A_col_ids, A_data, v1_n1_edge_index);
            int v1_n2;
            float v1_n2_edge_cost, v2_edge_cost;
            if (v1_n1_edge_cost > tol)
            {
                for(int l2 = A_symm_row_offsets[v1_n1]; l2 < A_symm_row_offsets[v1_n1 + 1] && rep_edge_cost < -tol; ++l2)
                {
                    v1_n2 = A_symm_col_ids[l2];
                    bool connected = are_connected_by(v1_n1, v2, v1_n2, 
                                                    A_row_offsets, A_col_ids, A_data, A_num_rows,
                                                    v1_n2_edge_index, v2_edge_index,
                                                    v1_n2_edge_cost, v2_edge_cost);

                    if (connected)
                    {
                        float packing_value = min(-rep_edge_cost, 
                                                min(v1_n1_edge_cost, 
                                                    min(v1_n2_edge_cost, v2_edge_cost)));

                        rep_edge_cost += packing_value;
                        atomicAdd(&A_data[v1_n1_edge_index], -packing_value);
                        atomicAdd(&A_data[v1_n2_edge_index], -packing_value);
                        atomicAdd(&A_data[v2_edge_index], -packing_value);
                        if (A_data[v1_n1_edge_index] < 0 || A_data[v1_n2_edge_index] < 0 || A_data[v2_edge_index] < 0)
                        {// Undo:
                            rep_edge_cost -= packing_value;
                            atomicAdd(&A_data[v1_n1_edge_index], packing_value);
                            atomicAdd(&A_data[v1_n2_edge_index], packing_value);
                            atomicAdd(&A_data[v2_edge_index], packing_value);    
                        }
                        int3 tri1 = make_int3(v1, v1_n1, v2);
                        if (empty_tri_index[0] < max_triangles)
                            triangle_vertices[atomicAdd(empty_tri_index, 1)] = tri1;
                        int3 tri2 = make_int3(v1_n1, v1_n2, v2);
                        if (empty_tri_index[0] < max_triangles)
                            triangle_vertices[atomicAdd(empty_tri_index, 1)] = tri2;
            
                    }
                }
            }
        }
        A_data[rep_edge_index] = rep_edge_cost;
        __syncthreads();
    }
}

__global__ void pack_pentagons_parallel(const int num_rep_edges, 
    const int* const __restrict__ row_ids_rep, 
    const int* const __restrict__ col_ids_rep, 
    const int* const __restrict__ A_symm_row_offsets,
    const int* const __restrict__ A_symm_col_ids,
    const int* const __restrict__ A_row_offsets, // adjacency matrix of original directed graph.
    const int* const __restrict__ A_col_ids,
    float* __restrict__ A_data,
    const int first_rep_edge_index,
    const int A_num_rows)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for (int edge = tid + first_rep_edge_index; edge < first_rep_edge_index + num_rep_edges; edge += num_threads) 
    {
        int v1 = row_ids_rep[edge];
        int v2 = col_ids_rep[edge];
        int rep_edge_index = -1;
        float rep_edge_cost = get_CSR_value(v1, v2, A_num_rows, A_row_offsets, A_col_ids, A_data, rep_edge_index);
        assert(rep_edge_cost < tol);
        assert(rep_edge_index >= 0); // The repulsive edge must also be present in A.(row -> col).

        int v1_n1_edge_index, v1_n2_edge_index, v2_n1_edge_index, v2_edge_index; 
        // Searching for a path like: v1 -(v1_n1_edge_index)- v1_n1 -(v1_n2_edge_index)- v1_n2 -(v2_n1_edge_index)- v2_n1 -(v2_edge_index)-  v2.
        for(int l1 = A_symm_row_offsets[v1]; l1 < A_symm_row_offsets[v1 + 1] && rep_edge_cost < -tol; ++l1)
        {
            int v1_n1 = A_symm_col_ids[l1];
            float v1_n1_edge_cost = get_CSR_value_both_dir_geq_tol(v1, v1_n1, A_num_rows, A_row_offsets, A_col_ids, A_data, v1_n1_edge_index);
            if (v1_n1_edge_cost < tol)
                continue; 

            for(int l2 = A_symm_row_offsets[v2]; l2 < A_symm_row_offsets[v2 + 1] && rep_edge_cost < -tol; ++l2)
            {
                int v2_n1 = A_symm_col_ids[l2];
                float v2_edge_cost = get_CSR_value_both_dir_geq_tol(v2, v2_n1, A_num_rows, A_row_offsets, A_col_ids, A_data, v2_edge_index);
                if (v2_edge_cost < tol)
                    continue;

                for(int l3 = A_symm_row_offsets[v1_n1]; l3 < A_symm_row_offsets[v1_n1 + 1] && rep_edge_cost < -tol; ++l3)
                {
                    int v1_n2 = A_symm_col_ids[l3];
                    float v1_n2_edge_cost, v2_n1_edge_cost;
                    bool connected = are_connected_by(v1_n1, v2_n1, v1_n2, 
                                                    A_row_offsets, A_col_ids, A_data, A_num_rows,
                                                    v1_n2_edge_index, v2_n1_edge_index,
                                                    v1_n2_edge_cost, v2_n1_edge_cost);
                    
                    if (!connected)
                        continue;

                    float packing_value = min(-rep_edge_cost, 
                                            min(v1_n1_edge_cost, 
                                                min(v1_n2_edge_cost,
                                                    min(v2_n1_edge_cost, v2_edge_cost))));

                    rep_edge_cost += packing_value;
                    atomicAdd(&A_data[v1_n1_edge_index], -packing_value);
                    atomicAdd(&A_data[v1_n2_edge_index], -packing_value);
                    atomicAdd(&A_data[v2_n1_edge_index], -packing_value);
                    atomicAdd(&A_data[v2_edge_index], -packing_value);
                    if (A_data[v1_n1_edge_index] < 0 || A_data[v1_n2_edge_index] < 0 || A_data[v2_edge_index] < 0 || A_data[v2_n1_edge_index] < 0)
                    {// Undo:
                        rep_edge_cost -= packing_value;
                        atomicAdd(&A_data[v1_n1_edge_index], packing_value);
                        atomicAdd(&A_data[v1_n2_edge_index], packing_value);
                        atomicAdd(&A_data[v2_n1_edge_index], packing_value);
                        atomicAdd(&A_data[v2_edge_index], packing_value);    
                    }
                }
            }
        }
        A_data[rep_edge_index] = rep_edge_cost;
        __syncthreads();
    }
}

struct is_positive_edge
{
    __host__ __device__ bool operator()(const thrust::tuple<int,int,float>& t)
    {
        if(thrust::get<2>(t) > 0.0f)
            return true;
        else
            return false;
    }
};

std::tuple<dCOO, thrust::device_vector<int>, thrust::device_vector<int>, int> create_matrices(hipsparseHandle_t handle, const dCOO& A)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME
    
    // Partition edges into positive and negative.
    thrust::device_vector<int> row_ids_rep = A.get_row_ids();
    thrust::device_vector<int> col_ids_rep = A.get_col_ids();
    thrust::device_vector<float> costs = A.get_data();

    auto first = thrust::make_zip_iterator(thrust::make_tuple(row_ids_rep.begin(), col_ids_rep.begin(), costs.begin()));
    auto last = thrust::make_zip_iterator(thrust::make_tuple(row_ids_rep.end(), col_ids_rep.end(), costs.end()));

    auto first_negative = thrust::partition(first, last, is_positive_edge());
    const size_t nr_positive_edges = std::distance(first, first_negative);

    // Create symmetric adjacency matrix of positive edges.
    thrust::device_vector<int> pos_row_ids_symm, pos_col_ids_symm;
    thrust::device_vector<float> pos_costs_symm;
    dCOO A_pos;
    if (nr_positive_edges > 0)
    {
        std::tie(pos_row_ids_symm, pos_col_ids_symm, pos_costs_symm) = to_undirected(row_ids_rep.begin(), row_ids_rep.begin() + nr_positive_edges,
                                                                                    col_ids_rep.begin(), col_ids_rep.begin() + nr_positive_edges,
                                                                                    costs.begin(), costs.begin() + nr_positive_edges);
        A_pos = dCOO(handle, std::max(A.rows(), A.cols()), std::max(A.rows(), A.cols()),
                        pos_col_ids_symm.begin(), pos_col_ids_symm.end(),
                        pos_row_ids_symm.begin(), pos_row_ids_symm.end(), 
                        pos_costs_symm.begin(), pos_costs_symm.end());
    }
    return {A_pos, row_ids_rep, col_ids_rep, nr_positive_edges};
}

// A should be directed thus containing same number of elements as in original problem. Does packing in-place on A.
double parallel_small_cycle_packing_cuda(hipsparseHandle_t handle, dCOO& A, const int max_tries_triangles, const int max_tries_quads)
{
    MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME;

    int num_nodes = A.rows();
    int num_edges = A.edges();
    double lb = get_lb(A.get_data());
    std::cout<<"Initial lb: "<<lb<<std::endl;

    // Make adjacency matrix and BFS search starting matrix.
    dCOO A_pos;
    thrust::device_vector<int> row_ids_rep, col_ids_rep;
    int nr_positive_edges;
    std::tie(A_pos, row_ids_rep, col_ids_rep, nr_positive_edges) = create_matrices(handle, A);
    if (nr_positive_edges == 0)
        return lb; 

    int num_rep_edges = num_edges - nr_positive_edges;
 
    thrust::device_vector<int> A_row_offsets = A.compute_row_offsets(handle);
    thrust::device_vector<int> A_pos_row_offsets = A_pos.compute_row_offsets(handle);

    int threadCount = 256;
    int blockCount = ceil(num_rep_edges / (float) threadCount);
    thrust::device_vector<int3> triangles(num_rep_edges * 10);
    thrust::device_vector<int> empty_tri_index(1, 0);

    for (int t = 0; t < max_tries_triangles; t++)
    {
        pack_triangles_parallel<<<blockCount, threadCount>>>(num_rep_edges, 
            thrust::raw_pointer_cast(row_ids_rep.data()), 
            thrust::raw_pointer_cast(col_ids_rep.data()), 
            thrust::raw_pointer_cast(A_pos_row_offsets.data()),
            A_pos.get_col_ids_ptr(),
            thrust::raw_pointer_cast(A_row_offsets.data()),
            A.get_col_ids_ptr(),
            A.get_writeable_data_ptr(),
            nr_positive_edges,
            A.rows(), 
            thrust::raw_pointer_cast(triangles.data()),
            thrust::raw_pointer_cast(empty_tri_index.data()),
            triangles.size());
        
        lb = get_lb(A.get_data());
        std::cout<<"packing triangles, itr: "<<t<<", lb: "<<lb<<", found # of triangles: "<<empty_tri_index[0]<<std::endl;
    }
    for (int t = 0; t < max_tries_quads; t++)
    {
        pack_quadrangles_parallel<<<blockCount, threadCount>>>(num_rep_edges, 
            thrust::raw_pointer_cast(row_ids_rep.data()), 
            thrust::raw_pointer_cast(col_ids_rep.data()), 
            thrust::raw_pointer_cast(A_pos_row_offsets.data()),
            A_pos.get_col_ids_ptr(),
            thrust::raw_pointer_cast(A_row_offsets.data()),
            A.get_col_ids_ptr(),
            A.get_writeable_data_ptr(),
            nr_positive_edges,
            A.rows(),
            thrust::raw_pointer_cast(triangles.data()),
            thrust::raw_pointer_cast(empty_tri_index.data()),
            triangles.size());
        
        lb = get_lb(A.get_data());
        std::cout<<"packing quadrangles, itr: "<<t<<", lb: "<<lb<<", found # of triangles: "<<empty_tri_index[0]<<std::endl;
    }

    return lb;
}

std::tuple<double, dCOO> parallel_small_cycle_packing_cuda(const std::vector<int>& i, const std::vector<int>& j, const std::vector<float>& costs, const int max_tries_triangles, const int max_tries_quads)
{
    const int cuda_device = get_cuda_device();
    hipSetDevice(cuda_device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device);
    std::cout << "Going to use " << prop.name << " " << prop.major << "." << prop.minor << ", device number " << cuda_device << "\n";
    hipsparseHandle_t handle;
    checkCuSparseError(hipsparseCreate(&handle), "cusparse init failed");
    
    dCOO A(handle, 
        i.begin(), i.end(),
        j.begin(), j.end(), 
        costs.begin(), costs.end());
    
    double lb = parallel_small_cycle_packing_cuda(handle, A, max_tries_triangles, max_tries_quads);
    return {lb, A};
}

double parallel_small_cycle_packing_cuda_lower_bound(const std::vector<int>& i, const std::vector<int>& j, const std::vector<float>& costs, const int max_tries_triangles, const int max_tries_quads)
{
    dCOO A; 
    double lb; 
    std::tie(lb, A) = parallel_small_cycle_packing_cuda(i, j, costs, max_tries_triangles, max_tries_quads);
    return lb;
}

