#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "icp.h"
#include <thrust/reduce.h>
#include <stdio.h>
#include "time_measure_util.h"
#include<set>
static const float tol = 1e-3;

// https://stackoverflow.com/questions/62091548/atomiccas-for-bool-implementation
static __inline__ __device__ bool atomicCAS(bool *address, bool compare, bool val)
{
    unsigned long long addr = (unsigned long long)address;
    unsigned pos = addr & 3;  // byte position within the int
    int *int_addr = (int *)(addr - pos);  // int-aligned address
    int old = *int_addr, assumed, ival;

    bool current_value;

    do
    {
        current_value = (bool)(old & ((0xFFU) << (8 * pos)));

        if(current_value != compare) // If we expected that bool to be different, then
            break; // stop trying to update it and just return it's current value

        assumed = old;
        if(val)
            ival = old | (1 << (8 * pos));
        else
            ival = old & (~((0xFFU) << (8 * pos)));
        old = atomicCAS(int_addr, assumed, ival);
    } while(assumed != old);

    return current_value;
}

__global__ void initialize(const int num_edges, const int* const __restrict__ e_row_ids, const int* const __restrict__ e_col_ids, const float* const __restrict__ e_values, 
                        int* const __restrict__ v_dist, int* const __restrict__ v_seed_edge, int* const __restrict__ v_parent_edge, 
                        bool* const __restrict__ e_valid_seeds, const bool* const __restrict__ e_used, 
                        bool* const __restrict__ still_running)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;

    for (int edge = tid; edge < num_edges; edge += num_threads) 
    {
        // printf("Hello from block %d, thread %d tid %d edge %d\n", blockIdx.x, threadIdx.x, tid, edge);
        if (e_values[edge] < -tol && !e_used[edge]) 
        {
            // printf("Neg edge block %d, thread %d, edge %d\n", blockIdx.x, threadIdx.x, edge);
            const int from_vertex = e_row_ids[edge];
            v_dist[from_vertex] = 0;
            
            // If from_vertex is part of more than one negative edge, then make it part of the edge with highest index
            atomicMax(&v_seed_edge[from_vertex], edge); 
            if (v_seed_edge[from_vertex] == edge) // winner thread.
            {
                // printf("Hello from block %d, thread %d, edge %d, propagating\n", blockIdx.x, threadIdx.x, edge);
                v_parent_edge[from_vertex] = edge;
                //TODO: Size of following array can be reduced to only contain negative edge indices.
                e_valid_seeds[edge] = true;
                *still_running = true;
            }
        }
        __syncthreads();
    }
}

__device__ void propagate(const int itr, const int edge, const int src_v, const int dst_v, const int src_seed_edge, const int dst_seed_edge, const float e_value, 
                        int* const __restrict__ v_seed_edge, int* const __restrict__ v_dist, int* const __restrict__ v_parent_edge, 
                        bool* const __restrict__ e_valid_seeds, const bool* const __restrict__ e_used, 
                        bool* const __restrict__ still_running)
{

    if (!e_valid_seeds[src_seed_edge]) // || e_used[dst_seed_edge])
        return; // Either the path is cut-off in which case expanding is useless, or the path was already explored and cycle was found in previous episode.

    atomicMax(&v_seed_edge[dst_v], src_seed_edge);

    if (src_seed_edge == v_seed_edge[dst_v]) // winner thread continues onward, 
    {
        e_valid_seeds[dst_seed_edge] = false; // Overridden by a higher priority path. 
        v_parent_edge[dst_v] = edge;
        v_dist[dst_v] = itr + 1;
        *still_running = true;
    }
}

__global__ void expand(const int iteration, const int num_edges, const int* const __restrict__ e_row_ids, const int* const __restrict__ e_col_ids, const float* const __restrict__ e_values, 
                    int* const __restrict__ v_dist, int* const __restrict__ v_seed_edge, int* const __restrict__ v_parent_edge, 
                    bool* const __restrict__ e_valid_seeds, const bool* const __restrict__ e_used, 
                    bool* const __restrict__ still_running)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int edge = tid; edge < num_edges; edge += num_threads) 
    {
        float e_value = e_values[edge];
        if (e_value > tol) 
        {
            const int n1 = e_row_ids[edge];
            const int n2 = e_col_ids[edge];
            const int n1_seed_edge = v_seed_edge[n1];
            const int n2_seed_edge = v_seed_edge[n2];
            
            // Propagate from n1 to n2 if:
            // 1. n1 is at frontier.
            // 2. n2 is unmarked (in which case n2_seed_edge would be -1) OR n2 is marked with a lower priority path.
            if (v_dist[n1] == iteration && n1_seed_edge > n2_seed_edge)
                propagate(iteration, edge, n1, n2, n1_seed_edge, n2_seed_edge, e_value, v_seed_edge, v_dist, v_parent_edge, e_valid_seeds, e_used, still_running);
            
            // Consider edge in opposite direction (n2 to n1) if previous attempt failed.
            else if (v_dist[n2] == iteration && n2_seed_edge > n1_seed_edge)
                propagate(iteration, edge, n2, n1, n2_seed_edge, n1_seed_edge, e_value, v_seed_edge, v_dist, v_parent_edge, e_valid_seeds, e_used, still_running);
            
            // When two ends of a cycle meet then n1_seed_edge = n2_seed_edge so no further propagation would happen.
        }
        __syncthreads();
    }
}


__device__ void print_cycle(int edge, int cycle_length, const int* const __restrict__ e_row_ids, const int* const __restrict__ e_col_ids, float* const __restrict__ e_values, 
    const int* const __restrict__ v_seed_edge, const int* const __restrict__ v_parent_edge, 
    bool* const __restrict__ e_valid_seeds, bool* const __restrict__ e_used)
{
    int to_vertex = e_col_ids[edge];
    int next_edge = v_parent_edge[to_vertex];
    for (int e = 0; e != cycle_length; ++e)
    {
        printf("edge: %d, next_edge: %d, seed_edge: %d, hop: %d, vertex: %d \n", edge, next_edge, v_seed_edge[to_vertex], e, to_vertex);
        assert(to_vertex != e_col_ids[next_edge] || to_vertex != e_row_ids[next_edge]);
        to_vertex = e_col_ids[next_edge] == to_vertex ? e_row_ids[next_edge] : e_col_ids[next_edge];
        next_edge = v_parent_edge[to_vertex];
    }

}

__global__ void reparameterize(int num_edges, int cycle_length, const int* const __restrict__ e_row_ids, const int* const __restrict__ e_col_ids, float* const __restrict__ e_values, 
    const int* const __restrict__ v_seed_edge, const int* const __restrict__ v_parent_edge, 
    bool* const __restrict__ e_valid_seeds, bool* const __restrict__ e_used, int* const __restrict__ num_cycles_packed)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;
    for (int edge = tid; edge < num_edges; edge += num_threads) 
    {
        const int from_vertex = e_row_ids[edge];
        int to_vertex = e_col_ids[edge];
        const int seed_edge = v_seed_edge[from_vertex];
        // seed is valid, edge is negative, both ends agree (valid cycle), not overriden, not already reparameterized.
        if (seed_edge >=0 && edge == seed_edge && seed_edge == v_seed_edge[to_vertex] && e_valid_seeds[seed_edge] && !e_used[seed_edge])
        {
            assert(e_values[seed_edge] < 0);
            bool old_value = atomicCAS(&e_valid_seeds[seed_edge], true, false);
            if (old_value) 
            {
                e_used[seed_edge] = true;
                bool invalid = false;
                float message = -e_values[seed_edge];
                assert(message >= 0);
                int next_edge = v_parent_edge[to_vertex];
                assert(next_edge >= 0);
                for (int e = 0; e != cycle_length - 1; ++e)
                {
                    //DEBUG:
                    // assert(v_seed_edge[to_vertex] == seed_edge);
                    if (v_seed_edge[to_vertex] != seed_edge || e_values[next_edge] <= 0 || next_edge == seed_edge)
                    {
                        invalid = true;
                        e_used[seed_edge] = false;
                        break;
                    }
                    
                    // assert(v_seed_edge[to_vertex] == seed_edge);
                    // assert(e_values[next_edge] >= 0);
                    message = min(e_values[next_edge], message);
                    to_vertex = e_col_ids[next_edge] == to_vertex ? e_row_ids[next_edge] : e_col_ids[next_edge];
                    next_edge = v_parent_edge[to_vertex];
                    assert(next_edge >= 0);
                }
                if (invalid || to_vertex != from_vertex)
                    continue;
                to_vertex = e_col_ids[edge];
                next_edge = v_parent_edge[to_vertex];
                assert(message >= 0);
                e_values[edge] += message; 
                atomicAdd(&num_cycles_packed[0], 1); // TODO: For debugging info, should remove during production.
                for (int e = 0; e != cycle_length - 1; ++e)
                {
                    assert(e_values[next_edge] >= 0);
                    assert(message >= 0);
                    if (e_values[next_edge] < message)
                    {
                        print_cycle(edge, cycle_length, e_row_ids, e_col_ids, e_values, 
                            v_seed_edge, v_parent_edge, 
                            e_valid_seeds, e_used);
                    }
                    assert(e_values[next_edge] >= message);

                    e_values[next_edge] -= message;
                    to_vertex = e_col_ids[next_edge] == to_vertex ? e_row_ids[next_edge] : e_col_ids[next_edge];
                    next_edge = v_parent_edge[to_vertex];
                }
            }
        }
        __syncthreads();
    }
}

std::set<int> find_cycle_edges(const int cycle_length, const int seed_edge, 
                            const std::vector<int>& row_ids, const std::vector<int>& col_ids, 
                            const std::vector<int>& v_parent_edge, const std::vector<int>& v_seed_edge)
{
    std::set<int> positive_edges;
    int start_vertex = row_ids[seed_edge];
    int end_vertex = col_ids[seed_edge];
    if (v_parent_edge[end_vertex] == seed_edge)
    {
        int temp = start_vertex;
        start_vertex = end_vertex;
        end_vertex = temp;
    }
    if (v_seed_edge[end_vertex] != seed_edge || v_parent_edge[start_vertex] != seed_edge)
        return positive_edges;

    for (int i = 0; i < cycle_length - 1; i++)
    {
        int next_edge = v_parent_edge[end_vertex];
        assert(next_edge != seed_edge);
        assert(positive_edges.find(next_edge) == positive_edges.end());
        positive_edges.insert(next_edge);
        end_vertex = col_ids[next_edge] == end_vertex ? row_ids[next_edge] : col_ids[next_edge];
    }
    assert(end_vertex == start_vertex);
    return positive_edges;
}

void check_detected_cycles(int cycle_length, const thrust::device_vector<int>& row_ids, const thrust::device_vector<int>& col_ids, const thrust::device_vector<float>& costs,
    const thrust::device_vector<float>& costs_reparam, const thrust::device_vector<int>& v_parent_edge, const thrust::device_vector<int>& v_dist, 
    const thrust::device_vector<bool>& e_used, const thrust::device_vector<bool>& prev_e_used, 
    const thrust::device_vector<int>& v_seed_edge, const thrust::device_vector<bool>& e_valid_seeds)
{
    std::vector<int> row_ids_h(row_ids.size());
    thrust::copy(row_ids.begin(), row_ids.end(), row_ids_h.begin());
    std::vector<int> col_ids_h(col_ids.size());
    thrust::copy(col_ids.begin(), col_ids.end(), col_ids_h.begin());
    std::vector<int> costs_h(costs.size());
    thrust::copy(costs.begin(), costs.end(), costs_h.begin());
    std::vector<int> costs_reparam_h(costs_reparam.size());
    thrust::copy(costs_reparam.begin(), costs_reparam.end(), costs_reparam_h.begin());
    std::vector<int> v_parent_edge_h(v_parent_edge.size());
    thrust::copy(v_parent_edge.begin(), v_parent_edge.end(), v_parent_edge_h.begin());
    std::vector<int> v_seed_edge_h(v_seed_edge.size());
    thrust::copy(v_seed_edge.begin(), v_seed_edge.end(), v_seed_edge_h.begin());
    std::vector<int> v_dist_h(v_dist.size());
    thrust::copy(v_dist.begin(), v_dist.end(), v_dist_h.begin());
    std::vector<bool> e_used_h(e_used.size());
    thrust::copy(e_used.begin(), e_used.end(), e_used_h.begin());
    std::vector<bool> prev_e_used_h(prev_e_used.size());
    thrust::copy(prev_e_used.begin(), prev_e_used.end(), prev_e_used_h.begin());
    std::vector<bool> e_valid_seeds_h(e_valid_seeds.size());
    thrust::copy(e_valid_seeds.begin(), e_valid_seeds.end(), e_valid_seeds_h.begin());
    std::vector<int> e_count(e_used_h.size(), 0);

    for (int e = 0; e < e_used_h.size(); e++)
    {
        if (prev_e_used_h[e] || costs_h[e] >= 0 || !e_used_h[e] || !e_valid_seeds_h[e])
            continue;
        
        std::set<int> pos_edges = find_cycle_edges(cycle_length, e, row_ids_h, col_ids_h, v_parent_edge_h, v_seed_edge_h);
        for (auto p: pos_edges)
        {
            assert(e_count[p] == 0);
            e_count[p]++;
        }
    }
}

// row_ids, col_ids, values should be directed thus containing same number of elements as in original problem.
std::tuple<thrust::device_vector<int>, thrust::device_vector<int>, thrust::device_vector<float>> parallel_cycle_packing_cuda(
    const thrust::device_vector<int>& row_ids, const thrust::device_vector<int>& col_ids, const thrust::device_vector<float>& costs,
    const int max_cycle_length, const int max_tries)
{
    // thrust::host_vector<float> costs_h = costs;
    MEASURE_FUNCTION_EXECUTION_TIME;

    int num_nodes = std::max(*thrust::max_element(row_ids.begin(), row_ids.end()), *thrust::max_element(col_ids.begin(), col_ids.end())) + 1;
    int num_edges = row_ids.size();
    thrust::device_vector<float> costs_reparam = costs;
    thrust::device_vector<int> v_seed_edge(num_nodes);
    thrust::device_vector<int> v_dist(num_nodes);
    thrust::device_vector<int> v_parent_edge(num_nodes);
    thrust::device_vector<bool> e_valid_seeds(num_edges);
    thrust::device_vector<bool> e_used(num_edges, false);
    thrust::device_vector<bool> still_running(1, false);
    thrust::device_vector<int> num_cycles_packed(1, 0);

    //DEBUG: 
    // thrust::device_vector<bool> prev_e_used(num_edges, false);

    int threadCount = 256;
    int blockCount = ceil(num_edges / (float) threadCount);
    int l = 3;

    int try_idx = 0;
    while(l <= max_cycle_length)
    {
        thrust::fill(thrust::device, v_seed_edge.begin(), v_seed_edge.end(), -1);
        thrust::fill(thrust::device, v_dist.begin(), v_dist.end(), -1);
        thrust::fill(thrust::device, v_parent_edge.begin(), v_parent_edge.end(), -1);
        thrust::fill(thrust::device, e_valid_seeds.begin(), e_valid_seeds.end(), false);
        thrust::fill(thrust::device, still_running.begin(), still_running.end(), false);

        initialize<<<blockCount, threadCount>>>(num_edges, 
                                            thrust::raw_pointer_cast(row_ids.data()), 
                                            thrust::raw_pointer_cast(col_ids.data()), 
                                            thrust::raw_pointer_cast(costs_reparam.data()), 
                                            thrust::raw_pointer_cast(v_dist.data()), 
                                            thrust::raw_pointer_cast(v_seed_edge.data()), 
                                            thrust::raw_pointer_cast(v_parent_edge.data()), 
                                            thrust::raw_pointer_cast(e_valid_seeds.data()), 
                                            thrust::raw_pointer_cast(e_used.data()), 
                                            thrust::raw_pointer_cast(still_running.data()));
        bool still_running_h = still_running[0];
        for (int itr = 0; itr < l - 1 && still_running_h; itr++)
        {   
            expand<<<blockCount, threadCount>>>(itr, num_edges, 
                                            thrust::raw_pointer_cast(row_ids.data()),
                                            thrust::raw_pointer_cast(col_ids.data()),
                                            thrust::raw_pointer_cast(costs_reparam.data()),
                                            thrust::raw_pointer_cast(v_dist.data()),
                                            thrust::raw_pointer_cast(v_seed_edge.data()),
                                            thrust::raw_pointer_cast(v_parent_edge.data()), 
                                            thrust::raw_pointer_cast(e_valid_seeds.data()),
                                            thrust::raw_pointer_cast(e_used.data()),
                                            thrust::raw_pointer_cast(still_running.data()));
            still_running_h = still_running[0];
        }

        try_idx++;
        if (!still_running_h || try_idx > max_tries)
        {
            thrust::fill(thrust::device, e_used.begin(), e_used.end(), false);
            thrust::fill(thrust::device, num_cycles_packed.begin(), num_cycles_packed.end(), 0);
            l++;
            try_idx = 0;
            continue;
        }
        reparameterize<<<blockCount, threadCount>>>(num_edges, l,
                                            thrust::raw_pointer_cast(row_ids.data()), 
                                            thrust::raw_pointer_cast(col_ids.data()), 
                                            thrust::raw_pointer_cast(costs_reparam.data()), 
                                            thrust::raw_pointer_cast(v_seed_edge.data()), 
                                            thrust::raw_pointer_cast(v_parent_edge.data()), 
                                            thrust::raw_pointer_cast(e_valid_seeds.data()), 
                                            thrust::raw_pointer_cast(e_used.data()), 
                                            thrust::raw_pointer_cast(num_cycles_packed.data()));

        thrust::transform(e_used.begin(), e_used.end(), e_valid_seeds.begin(), e_used.begin(), thrust::maximum<bool>());
        std::cout<<"cycle length: "<<l<<", cumulative # used -ive edges: "<<thrust::reduce(e_used.begin(), e_used.end(), 0)<<" cumulative # cycles packed: "<<num_cycles_packed[0]<<std::endl;

        // thrust::copy(costs_reparam.begin(), costs_reparam.end(), std::ostream_iterator<float>(std::cout, " "));
        // std::cout<<"\n";

        // check_detected_cycles(l, row_ids, col_ids, costs, costs_reparam, v_parent_edge, v_dist, e_used, prev_e_used, v_seed_edge, e_valid_seeds);
        // prev_e_used = e_used;
    }

    return {row_ids, col_ids, costs_reparam};
}